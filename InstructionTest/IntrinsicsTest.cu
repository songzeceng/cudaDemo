//
// Created by songzeceng on 2020/11/16.
//

#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void TestKernel(int *a) {
//    printf("%d\n", __popc(10));
//    printf("%d\n", __popc(9));
//    printf("%d\n", __popc(8));
//    printf("%d\n", __popc(11));
//    printf("%d\n", __popc(7));
//    printf("%d\n", __popc(15));
//    printf("%d\n", __popc(1));
//    printf("%d\n", __popc(0));
    // __popc()：返回二进制中1的数量

//    printf("%d\n", __ffs(0));
//    printf("%d\n", __ffs(1));
//    printf("%d\n", __ffs(7));
//    printf("%d\n", __ffs(8));
//    printf("%d\n", __ffs(9));
//    printf("%d\n", __ffs(10));
    // __ffs()：返回最低的1位，比如10(1010)返回2

    int x = atomicAdd(a, 5);
    printf("%d, %d\n", *a, x);
}

int main() {
    int *h_a = (int *)malloc(sizeof(int ));
    *h_a = 4;

    int *a;
    hipMalloc(&a, sizeof(int ));
    hipMemcpy(a, h_a, sizeof(int ), hipMemcpyHostToDevice);

    TestKernel<<<1, 1>>>(a);
    hipDeviceSynchronize();

    hipFree(a);
    free(h_a);
    return 0;
}
