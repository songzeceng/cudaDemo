//
// Created by songzeceng on 2020/11/16.
//

#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ int myAtomicAdd(int *address, int increment) {
    int expected = *address;
    int oldValue = atomicCAS(address, expected, expected + increment);
    // if value changed after *address and before atomicCAS, the oldValue will be different from expected.

    while (oldValue != expected) {
        expected = oldValue;
        oldValue = atomicCAS(address, expected, expected + increment);
    }
    return oldValue;
}

//__device__ float myAtomicAddFloat(float *ptr, float increment) {
//    int *ptr_int = reinterpret_cast<int *>(ptr);
//    // cannot get correct value in Pascal 6.1. We can use atomicAdd(float*, float) directly.
//
//    float current_value = *ptr;
//    int expected = __float2int_rn(current_value);
//    int desired = __float2int_rn(current_value + increment);
//
//    int oldValue = atomicCAS(ptr_int, expected, desired);
//    printf("Before CAS: current_value:%f, *ptr_int:%d, expected:%d, desired:%d, oldValue:%d\n",
//           current_value, *ptr_int, expected, desired, oldValue);
//    while (oldValue != expected) {
//        expected = oldValue;
//        desired = __float2int_rn(__int2float_rn(oldValue) + increment);
//        oldValue = atomicCAS(ptr_int, expected, desired);
//    }
//    printf("After CAS: current_value:%f, expected:%d, desired:%d, oldValue:%d\n",
//           current_value, expected, desired, oldValue);
//    return __int2float_rn(oldValue);
//}

__global__ void TestKernel(int *a) {
//    printf("%d\n", __popc(10));
//    printf("%d\n", __popc(9));
//    printf("%d\n", __popc(8));
//    printf("%d\n", __popc(11));
//    printf("%d\n", __popc(7));
//    printf("%d\n", __popc(15));
//    printf("%d\n", __popc(1));
//    printf("%d\n", __popc(0));
    // __popc()：返回二进制中1的数量

//    printf("%d\n", __ffs(0));
//    printf("%d\n", __ffs(1));
//    printf("%d\n", __ffs(7));
//    printf("%d\n", __ffs(8));
//    printf("%d\n", __ffs(9));
//    printf("%d\n", __ffs(10));
    // __ffs()：返回最低的1位，比如10(1010)返回2

    int x = atomicAdd(a, 5);
    x = myAtomicAdd(a, 4);
//    printf("%d, %d\n", *a, x);
}

//__global__ void testKernel2(float* value, float increment) {
//    printf("%f\n", myAtomicAddFloat(value, increment));
//}

// Use nvcc --ptx -o IntrinsicsTest.ptx IntrinsicsTest.cu to get ptx file
__global__ void intrinsic(float* ptr) {
    *ptr = __powf(*ptr, 2.1f);
}

__global__ void standard(float* ptr) {
    *ptr = powf(*ptr, 2.1f);
}

__shared__ int a[1];

__global__ void unsafe(int* values_read) {
    int old = a[0];
    a[0] = old + 1;
    values_read[threadIdx.x] = old;
}

__global__ void atomic(int* values_read) {
    values_read[threadIdx.x] = atomicAdd(a, 1);
}

int main() {
    int *h_a = (int *)malloc(sizeof(int ));
    *h_a = 4;

    int *a;
    hipMalloc(&a, sizeof(int ));
    hipMemcpy(a, h_a, sizeof(int ), hipMemcpyHostToDevice);

    TestKernel<<<1, 10>>>(a);

    hipMemcpy(h_a, a, sizeof(int ), hipMemcpyDeviceToHost);

    printf("%d\n", *h_a);

    int thread_num = 10;
    int nBytes = thread_num * sizeof(int );
    int *values_read_h = (int *) malloc(nBytes);
    int *values_read_d;
    hipMalloc(&values_read_d, nBytes);

    atomic<<<1, 10>>>(values_read_d);
    hipMemcpy(values_read_h, values_read_d, nBytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < thread_num; i++) {
        printf("%d\t", values_read_h[i]);
    }
    printf("\n");

    memset(values_read_h, 0, nBytes);
    hipMemset(values_read_d, 0, nBytes);

    unsafe<<<1, 10>>>(values_read_d);
    hipMemcpy(values_read_h, values_read_d, nBytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < thread_num; i++) {
        printf("%d\t", values_read_h[i]);
    }
    printf("\n");

//    float *h_f = (float *)malloc(sizeof(float ));
//    *h_f = 4.0f;
//
//    float *f;
//    cudaMalloc(&f, sizeof(float ));
//    cudaMemcpy(f, h_f, sizeof(float ), cudaMemcpyHostToDevice);
//
//    testKernel2<<<1, 10>>>(f, 1.2);
//
//    cudaMemcpy(h_f, f, sizeof(float ), cudaMemcpyDeviceToHost);

//    printf("%f\n", *h_f);

    hipFree(a);
    hipFree(values_read_d);
    free(h_a);
    free(values_read_h);
    return 0;
}
