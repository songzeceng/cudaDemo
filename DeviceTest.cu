//
// Created by songzeceng on 2020/11/8.
//


#include <hip/hip_runtime.h>
#include <stdio.h>

struct __align__(8) {
  int x;
  int y;
} A;

struct __align__(16) {
    int x;
    int y;
    int z;
} B;

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",device, deviceProp.major, deviceProp.minor);
    }
    return 0;
}