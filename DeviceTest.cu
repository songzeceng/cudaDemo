//
// Created by songzeceng on 2020/11/8.
//

#include "DeviceTest.cuh"
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",device, deviceProp.major, deviceProp.minor);
    }
    return 0;
}