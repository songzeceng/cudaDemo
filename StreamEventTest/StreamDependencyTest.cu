//
// Created by root on 2020/11/23.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

#define N 256

int n_streams = 4;

__global__ void kernel_1() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_1\n", sum);
}

__global__ void kernel_2() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_2\n", sum);
}

__global__ void kernel_3() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_3\n", sum);
}

__global__ void kernel_4() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_4\n", sum);
}

int main() {
    hipStream_t *streams = (hipStream_t *) malloc(n_streams * sizeof(hipStream_t));
    for (int i = 0; i < n_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    float elapsed_time = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEvent_t *kernelEvent = (hipEvent_t *) malloc(n_streams * sizeof(hipEvent_t));
    for (int i = 0; i < n_streams; i++) {
        hipEventCreateWithFlags(&kernelEvent[i], hipEventDisableTiming);
    }

    dim3 block(1);
    dim3 grid(1);

    hipEventRecord(start);
    for (int i = 0; i < n_streams; i++) {
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();
        hipEventRecord(kernelEvent[i], streams[i]);
        hipStreamWaitEvent(streams[n_streams - 1], kernelEvent[i], 0); // The last stream waits for the target event to complete
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("time elapsed between start and stop: %.4f", elapsed_time);
}
