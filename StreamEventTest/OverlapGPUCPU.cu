//
// Created by root on 2020/11/23.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

#define NSTREAM 4

#define n_repeat 32

__global__ void kernel(float *g_data, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    g_data[idx] = g_data[idx] + n;
}

void CUDART_CB my_callback(hipStream_t stream, hipError_t status, void *data) {
    printf("callback from stream %d\n", *((int *) data));
}

int main() {
    int nElem = 256;
    int nBytes = nElem * sizeof(float);

    float *d_A;
    float *h_A;

    hipMalloc(&d_A, nBytes);
    hipHostAlloc((void **) &h_A, nBytes, hipHostMallocDefault);

    for (int i = 0; i < nElem; i++) {
        h_A[i] = i;
    }

    int iElem = nElem / NSTREAM;
    int iBytes = iElem * sizeof(float);
    hipStream_t *streams = (hipStream_t *) malloc(NSTREAM * sizeof(hipStream_t));
    for (int i = 0; i < NSTREAM; i++) {
        hipStreamCreate(&streams[i]);
    }

    dim3 block(1);
    dim3 grid(iElem);

    hipEvent_t stop;
    hipEventCreate(&stop);

    int stream_id[NSTREAM] = {0, 1, 2, 3};
    // Deep first
    for (int i = 0; i < NSTREAM; i++) {
        int ioffset = i * iElem;
        hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, streams[i]);
        kernel<<<grid, block, 0, streams[i]>>>(&d_A[ioffset], NSTREAM);
        hipMemcpyAsync(&h_A[ioffset], &d_A[ioffset], iBytes, hipMemcpyDeviceToHost, streams[i]);
        hipEventRecord(stop, streams[NSTREAM - 1]);
        hipStreamAddCallback(streams[i], my_callback, (void *) &stream_id[i], 0);
    }

    int count = 0;
    while (hipEventQuery(stop) == hipErrorNotReady) {
        // Event has not accomplished in the specified stream
        // cudaEventQuery cannot co-exists with timing using cudaEventElapsed
        count++;
    }

//    printf("count = %d\n", count);
//    for (int i = 0; i < nElem; i++) {
//        printf("%.2f\t", h_A[i]);
//    }

    return 0;
}
