//
// Created by root on 2020/11/23.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

#define N 256

int n_streams = 4;

__global__ void kernel_1() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_1\n", sum);
}

__global__ void kernel_2() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_2\n", sum);
}

__global__ void kernel_3() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_3\n", sum);
}

__global__ void kernel_4() {
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.2);
    }
//    printf("sum=%.4f in kernel_4\n", sum);
}

int main() {
    setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1); // set max_connections to 1
    printf("CUDA_DEVICE_MAX_CONNECTIONS: %s\n", getenv("CUDA_DEVICE_MAX_CONNECTIONS"));

    hipStream_t *streams = (hipStream_t *) malloc(n_streams * sizeof(hipStream_t));
    for (int i = 0; i < n_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    float elapsed_time = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(1);
    dim3 grid(1);

//    cudaEventRecord(start);
//    // Deep-first schedule 0.6253ms
//    for (int i = 0; i < n_streams; i++) {
//        kernel_1<<<grid, block, 0, streams[i]>>>();
//        kernel_2<<<grid, block, 0, streams[i]>>>();
//        kernel_3<<<grid, block, 0, streams[i]>>>();
////        kernel_3<<<grid, block, 0>>>();
//        kernel_4<<<grid, block, 0, streams[i]>>>();
//    }
//    cudaEventRecord(stop);
//
//    cudaEventSynchronize(stop);
//    cudaEventElapsedTime(&elapsed_time, start, stop);
//    printf("time elapsed between start and stop: %.4f\n", elapsed_time);

    hipEventRecord(start);
    // Breadth-first schedule 0.2397ms
    for (int i = 0; i < n_streams; i++) {
        kernel_1<<<grid, block, 0, streams[i]>>>();
    }
    for (int i = 0; i < n_streams; i++) {
        kernel_2<<<grid, block, 0, streams[i]>>>();
    }
    for (int i = 0; i < n_streams; i++) {
        kernel_3<<<grid, block, 0, streams[i]>>>();
    }
    for (int i = 0; i < n_streams; i++) {
        kernel_4<<<grid, block, 0, streams[i]>>>();
    }

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("time elapsed between start and stop: %.4f", elapsed_time);
}
