//
// Created by root on 2020/11/23.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

#define NSTREAM 4

#define n_repeat 32

__global__ void sumArrays(float *A, float *B, float *C, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        for (int  i = 0; i < n_repeat; i++) {
            C[idx] = A[idx] + B[idx];
        }
    }
}

int main() {
    int nElem = 256;
    int nBytes = nElem * sizeof(float);

    float *gpu_ref;
    float *host_ref;
    float *d_A;
    float *d_B;
    float *h_A;
    float *h_B;

    hipHostAlloc((void **) &host_ref, nBytes, hipHostMallocDefault); // Async copy needs pinned host memory
    hipMalloc(&gpu_ref, nBytes);
    hipMalloc(&d_A, nBytes);
    hipMalloc(&d_B, nBytes);
    hipHostAlloc((void **) &h_A, nBytes, hipHostMallocDefault);
    hipHostAlloc((void **) &h_B, nBytes, hipHostMallocDefault);

    for (int i = 0; i < nElem; i++) {
        h_A[i] = i;
        h_B[i] = i + 1;
    }

    int iElem = nElem / NSTREAM;
    int iBytes = iElem * sizeof(float );
    hipStream_t *streams = (hipStream_t *) malloc(NSTREAM * sizeof(hipStream_t));
    for (int i = 0; i < NSTREAM; i++) {
        hipStreamCreate(&streams[i]);
    }

    dim3 block(1);
    dim3 grid(iElem);

//    // Deep first
//    for (int i = 0; i < NSTREAM; i++) {
//        int ioffset = i * iElem;
//        cudaMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, cudaMemcpyHostToDevice, streams[i]);
//        cudaMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, cudaMemcpyHostToDevice, streams[i]);
//        sumArrays<<<grid, block, 0, streams[i]>>>(&d_A[ioffset], &d_B[ioffset], &gpu_ref[ioffset], iElem);
//        cudaMemcpyAsync(&host_ref[ioffset], &gpu_ref[ioffset], iBytes, cudaMemcpyDeviceToHost, streams[i]);
//    }

    // Breadth first
    for (int i = 0; i < NSTREAM; i++) {
        int ioffset = i * iElem;
        hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, streams[i]);
    }
    for (int i = 0; i < NSTREAM; i++) {
        int ioffset = i * iElem;
        hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, hipMemcpyHostToDevice, streams[i]);
    }
    for (int i = 0; i < NSTREAM; i++) {
        int ioffset = i * iElem;
        sumArrays<<<grid, block, 0, streams[i]>>>(&d_A[ioffset], &d_B[ioffset], &gpu_ref[ioffset], iElem);
    }
    for (int i = 0; i < NSTREAM; i++) {
        int ioffset = i * iElem;
        hipMemcpyAsync(&host_ref[ioffset], &gpu_ref[ioffset], iBytes, hipMemcpyDeviceToHost, streams[i]);
    }

    for (int i = 0; i < nElem; i++) {
        printf("%.2f\t", host_ref[i]);
    }

    return 0;
}
