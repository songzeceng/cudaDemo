//
// Created by songzeceng on 2020/11/11.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

void getOccupancy();

__global__ void MyKernel(int* a, int* b, int* c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    c[idx] = a[idx] + b[idx];
}

__global__ void MyKernel2(int* a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        a[idx] = a[idx] + 3;
    }
}

int main() {
//    getOccupancy();

    int blockSize, minGridSize, gridSize;
    int size;

    scanf("%d", &size);

    int *h_data = (int *) malloc(size * sizeof(int));
    int *d_data;

    for (int i = 0; i < size; i++) {
        h_data[i] = i * i;
    }
    hipMalloc(&d_data, size * sizeof(int ));
    hipMemcpy(d_data, h_data, size * sizeof(int ), hipMemcpyHostToDevice);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void *)MyKernel2, 0, size);

    gridSize = (size + blockSize - 1) / blockSize;
    MyKernel2<<<gridSize, blockSize>>>(d_data, size);

    hipMemcpy(h_data, d_data, size * sizeof(int ), hipMemcpyDeviceToHost);
    for (int i = 0; i < size; i++) {
       printf("%d\n", h_data[i]);
    }

    hipFree(d_data);
    free(h_data);
    return 0;
}

void getOccupancy() {
    int numBlocks;
    int blockSize = 32;
    int device = 0, activeWraps, maxWarps;
    hipDeviceProp_t prop;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, MyKernel, blockSize, 0);

    activeWraps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    printf("Occupancy: %.3f\n", activeWraps / (double ) maxWarps);
}
