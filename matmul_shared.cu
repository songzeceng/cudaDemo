#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

typedef struct {
    int width;
    int height;
    int stride; // width of sub matrix
    float* data;
} Matrix;

#define BLOCK_SIZE 2

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

__device__ float GetElement(const Matrix A, int row, int col) {
    return A.data[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {
    A.data[row * A.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
    Matrix sub;

    sub.width = BLOCK_SIZE;
    sub.height = BLOCK_SIZE;
    sub.stride = A.stride;
    sub.data = &A.data[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return sub;
}

void showMatrix(Matrix m);

void matrixMultimal();

void MatMaul(const Matrix A, const Matrix B, Matrix C) {
    Matrix dA;
    dA.height = A.height;
    dA.width = A.width;
    dA.stride = A.width;
    size_t size = dA.width * dA.height * sizeof(float);
    hipMalloc(&dA.data, size);
    hipMemcpy(dA.data, A.data, size, hipMemcpyHostToDevice);

    Matrix dB;
    dB.height = B.height;
    dB.width = B.width;
    dB.stride = B.width;
    size = dB.width * dB.height * sizeof(float);
    hipMalloc(&dB.data, size);
    hipMemcpy(dB.data, B.data, size, hipMemcpyHostToDevice);

    Matrix dC;
    dC.height = C.height;
    dC.width = C.width;
    dC.stride = C.width;
    size = dC.width * dC.height * sizeof(float);
    hipMalloc(&dC.data, size);
//    cudaMemcpy(dC.data, C.data, size, cudaMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

    MatMulKernel<<<dimGrid, dimBlock>>>(dA, dB, dC);

    hipMemcpy(C.data, dC.data, size, hipMemcpyDeviceToHost);

    hipFree(dA.data);
    hipFree(dB.data);
    hipFree(dC.data);

    showMatrix(C);
}

__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    float value = 0;
    int row = threadIdx.y;
    int col = threadIdx.x;

    for (int i = 0; i < A.width / BLOCK_SIZE; i++) {
        Matrix Asub = GetSubMatrix(A, blockRow, i);
        Matrix Bsub = GetSubMatrix(B, i, blockCol);

        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE + 1];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE + 1];

        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        __syncthreads();

        for (int j = 0; j < BLOCK_SIZE; j++) {
            value += As[row][j] * Bs[j][col];
        }

        __syncthreads();
    }

    SetElement(Csub, row, col, value);
}

void showMatrix(Matrix m) {
    cout << "size(" << m.height << ", " << m.width << ")" << endl;
    for (int i = 0; i < m.height; i++) {
        for (int j = 0; j < m.width; j++) {
            cout << m.data[i * m.width + j] << "\t";
        }
        cout << endl;
    }
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
    }

    matrixMultimal();

    return 0;
}

void matrixMultimal() {
    Matrix A;
    Matrix B;
    Matrix C;

    A.height = 4;
    A.width = 4;
    A.stride = BLOCK_SIZE;
    A.data = (float *)malloc(A.width * A.height * sizeof(float));
    for (int i = 0; i < A.height * A.width; i++) {
        A.data[i] = i + 1;
    }

    B.height = 4;
    B.width = 4;
    B.stride = BLOCK_SIZE;
    B.data = (float *)malloc(B.width * B.height * sizeof(float));
    for (int i = 0; i < B.height * B.width; i++) {
        B.data[i] = 2 * i;
    }

    C.width = A.height;
    C.height = B.width;
    C.stride = BLOCK_SIZE;
    C.data = (float *)malloc(C.width * C.height * sizeof(float));

    showMatrix(A);
    showMatrix(B);

    MatMaul(A, B, C);
}
