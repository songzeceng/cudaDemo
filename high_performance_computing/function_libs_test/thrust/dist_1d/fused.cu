#include "hip/hip_runtime.h"
//
// Created by root on 2020/12/3.
//
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "thrust/iterator/counting_iterator.h"
#include "thrust/transform.h"
#include "math.h"
#include "stdio.h"

#define N 64

struct DistanceFrom {
    float mRef;
    int mN;

    DistanceFrom(float ref, int n) : mRef(ref), mN{n} {}

    __host__ __device__ float operator()(const float &x) {
        float scaledX = x / (mN - 1);
        return std::sqrt((scaledX - mRef) * (scaledX - mRef));
    }
};

int main() {
    float ref = 0.5f;

    thrust::device_vector<float> dvec_dist(N);
    thrust::transform(thrust::counting_iterator<float>(0), thrust::counting_iterator<float>(N), dvec_dist.begin(),
                      DistanceFrom(ref, N));
    // instantiate function object DistanceFrom.

    thrust::host_vector<float> h_dist(N);
    h_dist = dvec_dist;
    for (int i = 0; i < N; i++) {
        printf("x = %.3f, dist = %.3f\n", 1.f * i / (N - 1), h_dist[i]);
    }

    return 0;
}
