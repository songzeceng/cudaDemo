#include "hip/hip_runtime.h"
//
// Created by root on 2020/12/3.
//

#include "thrust/device_vector.h"
#include "thrust/inner_product.h"
#include "math.h"
#include "stdio.h"

#define N (1024 * 1024)

int main() {
    thrust::device_vector<float> d_vec(N, 1.2f);
    float norm = sqrt(thrust::inner_product(d_vec.begin(), d_vec.end(), d_vec.begin(), 0.0f));
    // parameters: begin and end of first vector, begin of second vector(the end of it is determined by the range of the first vector) and result init value
    printf("norm = %.2f\n", norm);
    return 0;
}