#include "hip/hip_runtime.h"
//
// Created by root on 2020/12/3.
//

#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/generate.h"
#include "thrust/count.h"
#include "stdio.h"

#define N (1 << 20)

// nvcc lambda.cu -o lambda --expt-extended-lambda
int main() {
    thrust::host_vector<float> h_x(N), h_y(N);

    thrust::generate(h_x.begin(), h_x.end(), rand);
    thrust::generate(h_y.begin(), h_y.end(), rand);

    thrust::device_vector<float> d_x = h_x;
    thrust::device_vector<float> d_y = h_y;

    int insideCount = thrust::count_if(thrust::make_zip_iterator(
            thrust::make_tuple(d_x.begin(), d_y.begin())), thrust::make_zip_iterator(
            thrust::make_tuple(d_x.end(), d_y.end())), []
    __device__(thrust::tuple < float, float > pair)
    {
        return (pow(thrust::get<0>(pair) / RAND_MAX, 2) + pow(thrust::get<1>(pair) / RAND_MAX, 2)) < 1.f;
    });

    printf("pi = %f\n", insideCount * 4.f / N);

    return 0;
}