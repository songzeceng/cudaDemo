#include "hip/hip_runtime.h"
//
// Created by root on 2020/12/3.
//

#include "hiprand.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/generate.h"
#include "thrust/count.h"
#include "stdio.h"

#define N (1 << 20)

//  nvcc pi_thust_curand.cu -o pi_thust_curand --expt-extended-lambda -lcurand
int main() {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 44559);

    thrust::device_vector<float> d_x(N);
    thrust::device_vector<float> d_y(N);

    float *p_x = thrust::raw_pointer_cast(&d_x[0]);
    float *p_y = thrust::raw_pointer_cast(&d_y[0]);
    hiprandGenerateUniform(gen, p_x, N);
    hiprandGenerateUniform(gen, p_y, N);
    hiprandDestroyGenerator(gen);

    int insideCount = thrust::count_if(thrust::make_zip_iterator(
            thrust::make_tuple(d_x.begin(), d_y.begin())), thrust::make_zip_iterator(
            thrust::make_tuple(d_x.end(), d_y.end())), []
    __device__(thrust::tuple < float, float > pair)
    {
        return (pow(thrust::get<0>(pair), 2) + pow(thrust::get<1>(pair), 2)) < 1.f;
    });

    printf("pi = %f\n", insideCount * 4.f / N);

    return 0;
}