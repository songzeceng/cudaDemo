//
// Created by songzeceng on 2020/11/26.
//
#include "stdio.h"
#include "hip/hip_runtime.h"

#define N 64
#define TPB 32

__device__ float scale(int i, int n) {
    return ((float ) i) / (n - 1);
}

__device__ float distance(float x1, float x2) {
    return sqrt((x2 - x1) * (x2 - x1));
}

__global__ void distanceKernel(float *d_out, float ref, int len) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float x = scale(i, len);
    d_out[i] = distance(x, ref);
}

int main() {
    float ref = 0.5f;

    float *d_out;
    float *h_out = (float *) malloc(N * sizeof(float ));
    hipMalloc(&d_out, N * sizeof(float ));

    distanceKernel<<<N / TPB, TPB>>>(d_out, ref, N);

    hipMemcpy(h_out, d_out, N * sizeof(float ), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%.2f\t", h_out[i]);
    }
    printf("\n");

    free(h_out);
    hipFree(d_out);

    return 0;
}
