//
// Created by root on 2020/11/26.
//

#include "hip/hip_runtime.h"
#include "stdio.h"

#define W 50
#define H 50
#define TX 32
#define TY 32

__device__ char clip(int n) {
    return n < 0 ? 0 : n > 255 ? 255 : n;
}

__global__ void distanceKernel(uchar4 *d_out, int w, int h, float2 pos) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = x + y * w;

    if (x >= w || y >= h) {
        return;
    }

    int d = sqrtf((x - pos.x) * (x - pos.x) + (y - pos.y) * (y - pos.y));
    char intensity = clip(255 - d);

    d_out[idx].x = intensity;
    d_out[idx].y = intensity;
    d_out[idx].z = 0;
    d_out[idx].w = 255;
}

int main() {
    int size = W * H;
    int nBytes = size * sizeof(uchar4);
    uchar4 *out = (uchar4 *) malloc(nBytes);
    uchar4 *d_out;
    hipMalloc(&d_out, nBytes);

    float2 pos = {0.0f, 0.0f};
    dim3 block(TX, TY);
    dim3 grid((W + TX - 1) / TX, (H + TY - 1) / TY);

    distanceKernel<<<grid, block>>>(d_out, W, H, pos);
    hipMemcpy(out, d_out, nBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            printf("(%d, %d, %d, %d)\t",
                   out[i * W + j].x, out[i * W + j].y, out[i * W + j].z, out[i * W + j].w);
        }
        printf("\n");
    }

    free(out);
    hipFree(d_out);

    return 0;
}