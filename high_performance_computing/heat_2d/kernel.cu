#include "hip/hip_runtime.h"
//
// Created by root on 2020/11/26.
//

#include "kernel.h"
#include "hip/hip_runtime.h"

#define TX 32
#define TY 32
#define RAD 1

int divUp(int a, int b) {
    return (a + b - 1) / b;
}

__device__ char clip(int n) {
    return n < 0 ? 0 : n > 255 ? 255 : n;
}

__device__ int idxClip(int idx, int idxMax) {
    return idx > idxMax - 1 ? (idxMax - 1) : (idx < 0 ? 0 : idx);
}

__device__ int flatten(int col, int row, int width, int height) {
    return idxClip(col, width) + idxClip(row, height) * width;
}

__global__ void resetKernel(float *d_temp, int w, int h, BC bc) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col >= w || row >= h) {
        return;
    }
    d_temp[row * w + col] = bc.t_a;
}

__global__ void tempKernel(uchar4 *d_out, float *d_temp, int w, int h, BC bc) {
    extern __shared__ float s_in[];
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col >= w || row >= h) {
        return;
    }

    int idx = flatten(col, row, w, h);
    int s_w = blockDim.x + 2 * RAD;
    int s_h = blockDim.y + 2 * RAD;
    int s_col = threadIdx.x + RAD;
    int s_row = threadIdx.y + RAD;
    int s_idx = flatten(s_col, s_row, s_w, s_h);

    d_out[idx].x = 0;
    d_out[idx].y = 0;
    d_out[idx].z = 0;
    d_out[idx].w = 255;

    s_in[s_idx] = d_temp[idx];
    if (threadIdx.x < RAD) {
        s_in[flatten(s_col - RAD, s_row, s_w, s_h)] = d_temp[flatten(col - RAD, row, w, h)];
        s_in[flatten(s_col + blockDim.x, s_row, s_w, s_h)] = d_temp[flatten(col + blockDim.x, row, w, h)];
    }
    if (threadIdx.y < RAD) {
        s_in[flatten(s_col, s_row - RAD, s_w, s_h)] = d_temp[flatten(col, row - RAD, w, h)];
        s_in[flatten(s_col, s_row + blockDim.y, s_w, s_h)] = d_temp[flatten(col, row + blockDim.y, w, h)];
    }

    float dSq = (col - bc.x) * (col - bc.x) + (row - bc.y) * (row - bc.y);
    if (dSq < bc.rad * bc.rad) {
        d_temp[idx] = bc.t_s;
    }
    if (col == 0 || col == w - 1 || row == 0 || col + row < bc.chamfer || col - row > w - bc.chamfer) {
        d_temp[idx] = bc.t_a;
    }
    if (row == h - 1) {
        d_temp[idx] = bc.t_g;
        return;
    }
    __syncthreads();
    float temp = 0.25f * (s_in[flatten(s_col - 1, s_row, s_w, s_h)] +
            s_in[flatten(s_col + 1, s_row, s_w, s_h)] +
            s_in[flatten(s_col, s_row - 1, s_w, s_h)] +
            s_in[flatten(s_col, s_row + 1, s_w, s_h)]);
    d_temp[idx] = temp;
    char intensity = clip((int ) temp);
    d_out[idx].x = intensity;
    d_out[idx].z = 255 - intensity;
}

void kernelLauncher(uchar4 *d_out, float *d_temp, int w, int h, BC bc) {
    dim3 block(TX, TY);
    dim3 grid(divUp(w, TX), divUp(h, TY));
    size_t smSz = (TX + 2 * RAD) * (TY + 2 * RAD) * sizeof(float );

    tempKernel<<<grid, block, smSz>>>(d_out, d_temp, w, h, bc);
}

void resetTemperature(float *d_temp, int w, int h, BC bc) {
    dim3 block(TX, TY);
    dim3 grid(divUp(w, TX), divUp(h, TY));

    resetKernel<<<grid, block>>>(d_temp, w, h, bc);
}