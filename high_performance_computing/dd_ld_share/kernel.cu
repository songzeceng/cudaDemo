#include "hip/hip_runtime.h"
//
// Created by songzeceng on 2020/11/26.
//

#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

#define TPB 128
#define RAD 1

__global__ void ddKernel(float *d_out, float *d_in, int size, float h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) {
        return;
    }

    int s_idx = threadIdx.x;
    extern __shared__ float s_in[];

    s_in[s_idx] = d_in[i];

    __syncthreads();
    if (threadIdx.x > 0) {
        float value = (s_in[s_idx - 1] - 2.f * s_in[s_idx] + s_in[s_idx + 1]) / (h * h);
        d_out[i] = value;
    }
}

void ddParallel(float *out, float *in, int n, float h) {
    float *d_in, *d_out;
    int nBytes = n * sizeof(float );
    hipMalloc(&d_in, nBytes);
    hipMalloc(&d_out, nBytes);
    hipMemcpy(d_in, in, nBytes, hipMemcpyHostToDevice);

    ddKernel<<<(n + TPB - 1) / TPB, TPB, (TPB + RAD) * sizeof(float )>>>(d_out, d_in, n, h);

    hipMemcpy(out, d_out, nBytes, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}