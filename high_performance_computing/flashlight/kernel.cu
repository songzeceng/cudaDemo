#include "hip/hip_runtime.h"
//
// Created by root on 2020/11/26.
//

#include "kernel.h"
#include "hip/hip_runtime.h"

#define TX 32
#define TY 32

__device__ char clip(int n) {
    return n < 0 ? 0 : n > 255 ? 255 : n;
}

__global__ void distanceKernel(uchar4 *d_out, int w, int h, int2 pos) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = y * w  + x;

    if (x >= w || y >= h) {
        return;
    }

    int d = sqrtf((x - pos.x) * (x - pos.x) + (y - pos.y) * (y - pos.y));
    char intensity = clip(255 - d);

    d_out[idx].x = intensity;
    d_out[idx].y = intensity;
    d_out[idx].z = 0;
    d_out[idx].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, int2 pos) {
    dim3 block(TX, TY);
    dim3 grid((w + TX - 1) / TX, (h + TY - 1) / TY);

    distanceKernel<<<grid, block>>>(d_out, w, h, pos);
}