#include "hip/hip_runtime.h"
//
// Created by root on 2020/11/30.
//

#include "kernel.cuh"
#include "stdio.h"
#include <hip/hip_vector_types.h>

#define TPB 64

__global__ void centroidKernel(uchar4 *d_img, int *d_centroidCol, int *d_centroidRow, int *d_pixelCount,
                               int width, int height) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int s_idx = threadIdx.x;
    int row = idx / width;
    int col = idx % width;

    __shared__ uint4 s_img[TPB];

    if ((d_img[idx].x < 255 || d_img[idx].y < 255 || d_img[idx].z < 255) && idx < width * height) {
        s_img[s_idx].x = col;
        s_img[s_idx].y = row;
        s_img[s_idx].z = 1;
    } else {
        s_img[s_idx].x = 0;
        s_img[s_idx].y = 0;
        s_img[s_idx].z = 0;
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (s_idx < s) {
            s_img[s_idx] += s_img[s_idx + s];
            __syncthreads();
        }
    }

    if (s_idx == 0) {
        atomicAdd(d_centroidCol, s_img[0].x);
        atomicAdd(d_centroidRow, s_img[0].y);
        atomicAdd(d_pixelCount, s_img[0].z);
    }
}

void centroidParallel(uchar4 *img, int width, int height) {
    uchar4 *d_img = 0;
    int *d_centroidRow = 0, *d_centroidCol = 0, *d_pixelCount = 0;
    int centroidRow = 0, centroidCol = 0, pixelCount = 0;

    hipMalloc(&d_img, width * height * sizeof(uchar4));
    hipMemcpy(d_img, img, width * height * sizeof(uchar4), hipMemcpyHostToDevice);
    hipMalloc(&d_centroidRow, sizeof(int ));
    hipMalloc(&d_centroidCol, sizeof(int ));
    hipMalloc(&d_pixelCount, sizeof(int ));
    hipMemset(d_centroidRow, 0, sizeof(int ));
    hipMemset(d_centroidCol, 0, sizeof(int ));
    hipMemset(d_pixelCount, 0, sizeof(int ));

    centroidKernel<<<(width * height + TPB - 1) / TPB, TPB>>>(d_img, d_centroidCol, d_centroidRow,
                                                              d_pixelCount, width, height);
    hipMemcpy(&centroidCol, d_centroidCol, sizeof(int ), hipMemcpyDeviceToHost);
    hipMemcpy(&centroidRow, d_centroidRow, sizeof(int ), hipMemcpyDeviceToHost);
    hipMemcpy(&pixelCount, d_pixelCount, sizeof(int ), hipMemcpyDeviceToHost);

    centroidCol /= pixelCount;
    centroidRow /= pixelCount;
    printf("Centroid: (col: %d, row: %d) based on %d pixels\n", centroidCol, centroidRow, pixelCount);

    for (int col = 0; col < width; col++) {
        img[centroidRow * width + col].x = 255;
        img[centroidRow * width + col].y = 0;
        img[centroidRow * width + col].z = 0;
    }
    for (int row = 0; row < height; row++) {
        img[row * width + centroidCol].x = 255;
        img[row * width + centroidCol].y = 0;
        img[row * width + centroidCol].z = 0;
    }

    hipFree(d_img);
    hipFree(d_centroidRow);
    hipFree(d_centroidCol);
    hipFree(d_pixelCount);
}