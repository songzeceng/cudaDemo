//
// Created by root on 2020/12/1.
//

#include "stdio.h"
#include "stdlib.h"
#include "hip/hip_runtime.h"

#define W 32
#define H 32
#define D 32
#define TX 8
#define TY 8
#define TZ 8

int divUp(int a, int b) {
    return (a + b - 1) / b;
}

__device__ float distance(int c, int r, int s, float3 pos) {
    return sqrtf((c - pos.x) * (c - pos.x) + (r - pos.y) * (r - pos.y) + (s - pos.z) * (s - pos.z));
}

__global__ void distanceKernel(float *d_out, int w, int h, int d, float3 pos) {
    int c = blockDim.x * blockIdx.x + threadIdx.x;
    int r = blockDim.y * blockIdx.y + threadIdx.y;
    int s = blockDim.z * blockIdx.z + threadIdx.z;
    int i = c + r * w + s * w * h;
    if (c >= w || r >= h || s >= d) {
        return;
    }
    d_out[i] = distance(c, r, s, pos);
}

int main() {
    float *out = (float *) malloc(W * H * D * sizeof(float ));
    float *d_out;
    hipMalloc(&d_out, W * H * D * sizeof(float ));

    float3 pos = {0.0f, 0.0f, 0.0f};
    dim3 block(TX, TY, TZ);
    dim3 grid(divUp(W, TX), divUp(H, TY), divUp(D, TZ));
    distanceKernel<<<grid, block>>>(d_out, W, H, D, pos);
    hipMemcpy(out, d_out, W * H * D * sizeof(float ), hipMemcpyDeviceToHost);
    hipFree(d_out);
    free(out);
    return 0;
}