//
// Created by songzeceng on 2020/11/26.
//

#include "hip/hip_runtime.h"
#include "stdio.h"

#define N 64
#define TPB 32

float scale(int i, int n) {
    return ((float ) i) / (n - 1);
}

__device__ float distance(float x1, float x2) {
    return sqrt((x2 - x1) * (x2 - x1));
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float x = d_in[i];
    d_out[i] = distance(x, ref);
}

int main() {
    float ref = 0.5f;
    float *in;
    float *out;

    hipMallocManaged(&in, N * sizeof(float ));
    hipMallocManaged(&out, N * sizeof(float ));

    for (int i = 0; i < N; ++i) {
        in[i] = scale(i, N);
    }

    distanceKernel<<<N / TPB, TPB>>>(out, in, ref);
    hipDeviceSynchronize();

    for (int i = 0; i < N; ++i) {
        printf("%.2f\t", out[i]);
    }
    printf("\n");

    hipFree(in);
    hipFree(out);
    return 0;
}