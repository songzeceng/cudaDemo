#include "hip/hip_runtime.h"
//
// Created by songzeceng on 2020/11/26.
//

#include "hip/hip_runtime.h"
#include "kernel.h"

#define TPB 32

__device__ float distance(float x1, float x2) {
    return sqrt((x2 - x1) * (x2 - x1));
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float x = d_in[i];
    d_out[i] = distance(x, ref);
}

void distanceArray(float *out, float *in, float ref, int len) {
    float *d_in;
    float *d_out;
    hipMalloc(&d_in, len * sizeof(float ));
    hipMalloc(&d_out, len * sizeof(float ));
    hipMemcpy(d_in, in, len * sizeof(float ), hipMemcpyHostToDevice);

    distanceKernel<<<len / TPB, TPB>>>(d_out, d_in, ref);

    hipMemcpy(out, d_out, len * sizeof(float ), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}