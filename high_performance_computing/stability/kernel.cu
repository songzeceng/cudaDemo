#include "hip/hip_runtime.h"
//
// Created by root on 2020/11/26.
//

#include "kernel.h"
#include "hip/hip_runtime.h"

#define TX 32
#define TY 32
#define LEN 5.f
#define TIME_STEP 0.005f
#define FINAL_TIME 10.f

__device__ float scale(int i, int w) {
    return 2 * LEN * (((1.f * i) / w) - 0.5f);
}

__device__ float f(float x, float y, float param, float sys) {
    if (sys == 1) {
        return x - 2 * param * y;
    }
    if (sys == 2) {
        return -x + param * (1 - x * x) * y;
    }
    return -x - 2 * param * y;
}

__device__ float2 euler(float x, float y, float dt, float tFinal, float param, float sys) {
    float dx = 0.f, dy = 0.f;
    for (float t = 0; t < tFinal; t += dt) {
        dx = dt * y;
        dy = dt * f(x, y , param, sys);
        x += dx;
        y += dy;
    }
    return make_float2(x, y);
}

__device__ char clip(float n) {
    return n < 0 ? 0 : n > 255 ? 255 : n;
}

__global__ void stabImageKernel(uchar4 *d_out, int w, int h, float p, int s) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = y * w  + x;

    if (x >= w || y >= h) {
        return;
    }

    float x0 = scale(x, w);
    float y0 = scale(y, h);
    float dist_0 = sqrt(x0 * x0 + y0 * y0);
    float2 pos = euler(x0, y0, TIME_STEP, FINAL_TIME, p, s);
    float dist_f = sqrt(pos.x * pos.x + pos.y * pos.y);
    float dist_r = dist_f / dist_0;

    d_out[idx].x = clip(dist_r * 255);
    d_out[idx].y = (x == w / 2 || y == h / 2) ? 255 : 0;
    d_out[idx].z = clip((1 / dist_r) * 255);
    d_out[idx].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, float p, int s) {
    dim3 block(TX, TY);
    dim3 grid((w + TX - 1) / TX, (h + TY - 1) / TY);

    stabImageKernel<<<grid, block>>>(d_out, w, h, p, s);
}