//
// Created by root on 2020/11/12.
//

#include "hip/hip_runtime.h"
#include "stdio.h"


__device__ int *m = NULL, *n = NULL;
__device__ volatile int *m_v = NULL, *n_v = NULL;
__device__ int A, B, A_v, B_v;

__device__ void write() {
//    m = 10;
//    n = 20;
//
//    m_v = 10;
//    n_v = 20;

    (*m)++;
    (*n)++;

    (*m_v)++;
    (*n_v)++;
}

__device__ void read() {
    A = *m;
    B = *n;

    A_v = *m_v;
    B_v = *n_v;
}

__global__ void testKernel(int *count) {
    if ((blockIdx.x * blockDim.x + threadIdx.x) == 0 && threadIdx.y == 0) {
        m = &A;
        n = &B;
        m_v = &A_v;
        n_v = &B_v;

        (*m) = 0;
        (*n) = 0;
        (*m_v) = 0;
        (*n_v) = 0;

//        printf("thread 0\n");
    } else {
//        printf("following thread\n");
    }
    write();

//    __shared__ int t;
//    t++;
//
//    printf("t = %d\n", t);
//    __threadfence();
//    __syncthreads();

    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = threadIdx.y + blockIdx.y * blockDim.y;
    *count = 1 + idx_x + idx_y * blockDim.x * gridDim.x;
    printf("count = %d\n", *count);

    read();

//    printf("A = %d, B = %d, A_v = %d, B_v = %d\n", A, B, A_v, B_v);
}

int *m_h = NULL, *n_h = NULL;
volatile int *m_v_h = NULL, *n_v_h = NULL;
int A_h, B_h, A_v_h, B_v_h;

void writeH() {
//    m = 10;
//    n = 20;
//
//    m_v = 10;
//    n_v = 20;

    (*m_h)++;
    (*n_h)++;

    (*m_v_h)++;
    (*n_v_h)++;
}

void readH() {
    A_h = *m_h;
    B_h = *n_h;

    A_v_h = *m_v_h;
    B_v_h = *n_v_h;
}

void testHost(int epoch) {
    if (epoch == 0) {
        m_h = &A_h;
        n_h = &B_h;
        m_v_h = &A_v_h;
        n_v_h = &B_v_h;

        (*m_h) = 0;
        (*n_h) = 0;
        (*m_v_h) = 0;
        (*n_v_h) = 0;
    }
    writeH();
    readH();

    printf("A_h = %d, B_h = %d, A_v_h = %d, B_v_h = %d\n", A_h, B_h, A_v_h, B_v_h);
}

int main() {
    int x = 10, y = 2;
    dim3 block(2, 2);
    dim3 grid((x + block.x - 1) / block.x, (y + block.y - 1) / block.y);

    int count = 0;
    int *countD;
    hipMalloc(&countD, sizeof(int));
    hipMemcpy(countD, &count, sizeof(count), hipMemcpyHostToDevice);
    testKernel<<<grid, block>>>(countD);
//    cudaError_t err = cudaDeviceSynchronize();
//    printf("result:%s\n", cudaGetErrorString(err));

    hipMemcpy(&count, countD, sizeof(count), hipMemcpyDeviceToHost);
//    printf("count = %d\n", count);

    hipMemcpy(&A_h, &A, sizeof(A), hipMemcpyDeviceToHost);
    hipMemcpy(&B_h, &B, sizeof(B), hipMemcpyDeviceToHost);
    hipMemcpy(&A_v_h, &A_v, sizeof(A_v), hipMemcpyDeviceToHost);
    hipMemcpy(&B_v_h, &B_v, sizeof(B_v), hipMemcpyDeviceToHost);

    printf("===============================\n");

    printf("A_h = %d, B_h = %d, A_v_h = %d, B_v_h = %d\n", A_h, B_h, A_v_h, B_v_h);

//    for (int i = 0; i < x * y; i++) {
//        testHost(i);
//    }

    return 0;
}