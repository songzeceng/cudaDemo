//
// Created by root on 2020/11/11.
//

#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void LocateThreadIdKernel() {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int z = blockDim.z * blockIdx.z + threadIdx.z;

//    printf("%d, %d. %d\n", threadIdx.x, threadIdx.y, threadIdx.z);

    printf("Thread coordinate: (%d, %d, %d)\n", x, y, z);
}

int main () {
    int x = 10, y = 15, z = 20;
    dim3 block(2, 3, 4);
    dim3 grid((x + block.x - 1) / block.x, (y + block.y - 1) / block.y, (z + block.z - 1) / block.z);
    LocateThreadIdKernel<<<grid, block>>>();
    hipDeviceSynchronize();

    return 0;
}