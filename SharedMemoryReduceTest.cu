//
// Created by root on 2020/11/19.
//

#include "stdio.h"
#include <hip/hip_runtime.h>

#define DIM 128

__global__ void reduceGmem(int *g_idata, int *g_odata, int n) {
    int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= n) {
        return;
    }

    // if current thread id is less than half of block dim, reduce in place
    if (blockDim.x >= 1024 && tid < 512) {
        idata[tid] += idata[tid + 512];
    }

    if (blockDim.x >= 512 && tid < 256) {
        idata[tid] += idata[tid + 256];
    }

    if (blockDim.x >= 256 && tid < 128) {
        idata[tid] += idata[tid + 128];
    }

    if (blockDim.x >= 128 && tid < 64) {
        idata[tid] += idata[tid + 64];
    }
    __syncthreads();

    // unrolling warp into the first thread of this warp
    if (tid < 32) {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32]; // I only applied block dim = 128, so tid + 64 has been reduced, but tid + 32 not
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result stored in thread 0 into output
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}

__global__ void reduceGmemUnrolling4(int *g_idata, int *g_odata, int n) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x; // one thread per block processes 4 data

    int *idata = g_idata + blockDim.x * blockIdx.x * 4;

    if (idx >= n) {
        return;
    }

    // process 4 data per thread
    int a = 0, b = 0, c = 0, d = 0;
    a = g_idata[idx];
    if (idx + blockDim.x < n) {
        b = g_idata[idx + blockDim.x];
    }
    if (idx + 2 * blockDim.x < n) {
        c = g_idata[idx + blockDim.x * 2];
    }
    if (idx + 3 * blockDim.x < n) {
        d = g_idata[idx + blockDim.x * 3];
    }
    g_idata[idx] = a + b + c + d;

    __syncthreads();

    if (blockDim.x >= 1024 && tid < 512) {
        idata[tid] += idata[tid + 512];
    }
    if (blockDim.x >= 512 && tid < 256) {
        idata[tid] += idata[tid + 256];
    }
    if (blockDim.x >= 256 && tid < 128) {
        idata[tid] += idata[tid + 128];
    }
    if (blockDim.x >= 128 && tid < 64) {
        idata[tid] += idata[tid + 64];
    }

    __syncthreads();

    if (tid < 32) {
        volatile int *s_vmem = idata;
        s_vmem[tid] += s_vmem[tid + 32];
        s_vmem[tid] += s_vmem[tid + 16];
        s_vmem[tid] += s_vmem[tid + 8];
        s_vmem[tid] += s_vmem[tid + 4];
        s_vmem[tid] += s_vmem[tid + 2];
        s_vmem[tid] += s_vmem[tid + 1];
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}

__global__ void reduceSMemUnrolling4(int *g_idata, int *g_odata, int n) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x * 4 + tid;

    if (idx >= n) {
        return;
    }

    extern __shared__ int smem[]; // I use dynamic shared memory to reduce data

    int a = 0, b = 0, c = 0, d = 0; // Each thread still processes 4 data
    a = g_idata[idx];
    if (idx + blockDim.x < n) {
        b = g_idata[idx + blockDim.x];
    }
    if (idx + 2 * blockDim.x < n) {
        c = g_idata[idx + 2 * blockDim.x];
    }
    if (idx + 3 * blockDim.x < n) {
        d = g_idata[idx + 3 * blockDim.x];
    }
    smem[tid] = a + b + c + d; // Save result of 4 data into shared memory
    __syncthreads();

    // Reduce data for block using shared memory
    if (blockDim.x >= 1024 && tid < 512) {
        smem[tid] += smem[tid + 512];
    }
    if (blockDim.x >= 512 && tid < 256) {
        smem[tid] += smem[tid + 256];
    }
    if (blockDim.x >= 256 && tid < 128) {
        smem[tid] += smem[tid + 128];
    }
    if (blockDim.x >= 128 && tid < 64) {
        smem[tid] += smem[tid + 64];
    }
    __syncthreads();

    if (tid < 32) {
        volatile int* s_vmem = smem;
        s_vmem[tid] += s_vmem[tid + 32];
        s_vmem[tid] += s_vmem[tid + 16];
        s_vmem[tid] += s_vmem[tid + 8];
        s_vmem[tid] += s_vmem[tid + 4];
        s_vmem[tid] += s_vmem[tid + 2];
        s_vmem[tid] += s_vmem[tid + 1];
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = smem[0];
    }
}

void test(int size) {
    int sum = 0;
    for (int i = 0; i < size; i++) {
        sum += i;
    }
    printf("Target is %d\n", sum);
}

int main() {
    int size = 1 << 22;
    int blockSize = DIM;

    test(size); // verify the result

    dim3 blockDim(blockSize);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x);
    printf("grid:(%d), block:(%d)\n", gridDim.x, blockDim.x);

    int nBytes = size * sizeof(int);
    int *h_idata = (int *) malloc(nBytes);
    int *h_odata = (int *) malloc(gridDim.x * sizeof(int));
    // Valid output per block is stored in the first thread of each block.
    // So the number of output to be added is equal to the grid dim

    int *d_odata;
    int *d_idata;

    hipMalloc(&d_idata, nBytes);
    hipMalloc(&d_odata, gridDim.x * sizeof(int));

    for (int i = 0; i < size; i++) {
        h_idata[i] = i;
    }

    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    reduceGmem<<<gridDim, blockDim>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();

    hipMemcpy(h_odata, d_odata, gridDim.x * sizeof(int), hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i = 0; i < gridDim.x; i++) {
        sum += h_odata[i];
    }
    printf("\n=========\n");
    printf("sum = %d\n", sum);

    memset(h_odata, 0, gridDim.x * sizeof(int));
    hipMemset(d_odata, 0, gridDim.x * sizeof(int));
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);

    dim3 gridDim_(gridDim.x / 4);
    reduceGmemUnrolling4<<<gridDim_, blockDim>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, gridDim_.x * sizeof(int), hipMemcpyDeviceToHost);
    sum = 0;
    for (int i = 0; i < gridDim_.x; i++) {
        sum += h_odata[i];
    }
    printf("\n=========\n");
    printf("sum = %d\n", sum);

    memset(h_odata, 0, gridDim.x * sizeof(int));
    hipMemset(d_odata, 0, gridDim.x * sizeof(int));
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);

    reduceSMemUnrolling4<<<gridDim_, blockDim, DIM * sizeof(int )>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, gridDim_.x * sizeof(int), hipMemcpyDeviceToHost);
    sum = 0;
    for (int i = 0; i < gridDim_.x; i++) {
        sum += h_odata[i];
    }
    printf("\n=========\n");
    printf("sum = %d\n", sum);

    hipFree(d_idata);
    hipFree(d_odata);
    free(h_idata);
    free(h_odata);

    return 0;
}