//
// Created by root on 2020/11/20.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

#define BDIMX 16
#define SEGM 4

#define DIM 32
#define SMEMDIM 4

__global__ void test_shuffle_broadcast(int *d_out, int *d_in, int srcLane) {
    int value = d_in[threadIdx.x];
    value = __shfl(value, srcLane, BDIMX);
    // change data with srcLane thread. cyclic is supported if necessary.
    // if there is a dependency, the smaller thread index executes earlier.Else executions will do in parallel.
    // when thread 0 is changing data with thread 1 and thread 1 is changing with thread 3, the former executes first
    d_out[threadIdx.x] = value;
}

__global__ void test_shuffle_up(int *d_out, int *d_in, int delta) {
    int value = d_in[threadIdx.x];
    value = __shfl_up(value, delta, BDIMX);
    d_out[threadIdx.x] = value;
}

__global__ void test_shuffle_down(int *d_out, int *d_in, int delta) {
    int value = d_in[threadIdx.x];
    value = __shfl_down(value, delta, BDIMX);
    d_out[threadIdx.x] = value;
}

__global__ void test_shuffle_cycle(int *d_out, int *d_in, int delta) {
    int value = d_in[threadIdx.x];
    int value_ = __shfl(value, threadIdx.x + delta, BDIMX);
    d_out[threadIdx.x] = value_;
}

__global__ void test_shuffle_butterfly(int *d_out, int *d_in, int delta) {
    int value = d_in[threadIdx.x];
    int value_ = __shfl_xor(value, delta, BDIMX);
    // change data with next thread
    // t0 with t1, t2 with t3, if no t5, data in t4 will be 0
    d_out[threadIdx.x] = value_;
}

__global__ void test_shuffle_xor_array(int *d_out, int *d_in, int mask) {
    int idx = threadIdx.x * SEGM;
    int value[SEGM];

    for (int i = 0; i < SEGM; i++) {
        value[i] = d_in[idx + i];
        value[i] = __shfl_xor(value[i], mask, SEGM);
        d_out[idx + i] = value[i];
    }
}

__inline__ __device__ void swap(int *value, int laneIdx, int mask, int firstIdx, int secondIdx) {
    bool pred = ((laneIdx % 2) == 0); // first thread per pair or not
    if (pred) {
        int temp = value[firstIdx];
        value[firstIdx] = value[secondIdx];
        value[secondIdx] = temp;
    }

    value[secondIdx] = __shfl_xor(value[secondIdx], mask, BDIMX);

    if (pred) {
        int temp = value[firstIdx];
        value[firstIdx] = value[secondIdx];
        value[secondIdx] = temp;
    }
}

__global__ void test_shuffle_warp(int *d_out, int *d_in, int mask, int firstIndex, int secondIndex) {
    int idx = threadIdx.x * SEGM;
    int value[SEGM];

    for (int i = 0; i < SEGM; i++) {
        value[i] = d_in[idx + i];
    }

    swap(value, threadIdx.x, mask, firstIndex, secondIndex);

    for (int i = 0; i < SEGM; i++) {
        d_out[idx + i] = value[i];
    }
}

__inline__ __device__ int warpReduce(int mySum) {
    mySum += __shfl_xor(mySum, 16);
    mySum += __shfl_xor(mySum, 8);
    mySum += __shfl_xor(mySum, 4);
    mySum += __shfl_xor(mySum, 2);
    mySum += __shfl_xor(mySum, 1);
    return mySum;
}

__global__ void reduceShuffle(int *g_idata, int *g_odata, int n) {
    __shared__ int smem[SMEMDIM];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) {
        return;
    }

    int mySum = g_idata[idx]; // current data

    int laneIdx = threadIdx.x % warpSize; // index in warp
    int warpIdx = threadIdx.x / warpSize; // warp index

    mySum = warpReduce(mySum); // reduce sum in a warp

    if (laneIdx == 0) {
        smem[warpIdx] = mySum; // if it`s the first thread in the warp, store warp sum into shared memory
    }

    __syncthreads();

    mySum = (threadIdx.x < SMEMDIM) ? smem[laneIdx] : 0; // get each warp`s sum

    if (warpIdx == 0) {
        mySum = warpReduce(mySum); // if it`s the first warp, reduce sums of each warp for the block
    }

    if (threadIdx.x == 0) {
        g_odata[blockIdx.x] = mySum; // store the block sum into the first thread of this block.
    }
}

int main() {
    int nx = BDIMX;
    int nBytes = nx * sizeof(int);

    int *h_in = (int *) malloc(nBytes);
    int *h_out = (int *) malloc(nBytes);

    for (int i = 0; i < nx; i++) {
        h_in[i] = i;
    }

    int *d_in;
    int *d_out;

    hipMalloc(&d_in, nBytes);
    hipMalloc(&d_out, nBytes);

    hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice);

    dim3 blockDim(BDIMX);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x);

    test_shuffle_broadcast<<<gridDim, blockDim>>>(d_out, d_in, 2);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);
    test_shuffle_up<<<gridDim, blockDim>>>(d_out, d_in, 2);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);
    test_shuffle_down<<<gridDim, blockDim>>>(d_out, d_in, 2);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);
    test_shuffle_cycle<<<gridDim, blockDim>>>(d_out, d_in, 2);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);
    test_shuffle_butterfly<<<gridDim, blockDim>>>(d_out, d_in, 1);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);
    test_shuffle_xor_array<<<1, BDIMX / SEGM>>>(d_out, d_in, 1);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);
    test_shuffle_warp<<<1, BDIMX / SEGM>>>(d_out, d_in, 1, 0, 3);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);
    blockDim.x = DIM;
    gridDim.x = (nx + blockDim.x - 1) / blockDim.x;
    reduceShuffle<<<gridDim, blockDim>>>(d_in, d_out, nx);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);

    printf("\n============\n");
    for (int i = 0; i < nx; i++) {
        printf("%d->", h_out[i]);
    }

    free(h_out);
    hipFree(d_out);
    free(h_in);
    hipFree(d_in);
}
