#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

typedef struct {
    int width;
    int height;
    float* data;
} Matrix;

#define BLOCK_SIZE 2

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void showMatrix(Matrix m);

void MatMaul(const Matrix A, const Matrix B, Matrix C) {
    Matrix dA;
    dA.height = A.height;
    dA.width = A.width;
    size_t size = dA.width * dA.height * sizeof(float);
    hipMalloc(&dA.data, size);
    hipMemcpy(dA.data, A.data, size, hipMemcpyHostToDevice);

    Matrix dB;
    dB.height = B.height;
    dB.width = B.width;
    size = dB.width * dB.height * sizeof(float);
    hipMalloc(&dB.data, size);
    hipMemcpy(dB.data, B.data, size, hipMemcpyHostToDevice);

    Matrix dC;
    dC.height = C.height;
    dC.width = C.width;
    size = dC.width * dC.height * sizeof(float);
    hipMalloc(&dC.data, size);
//    cudaMemcpy(dC.data, C.data, size, cudaMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

    MatMulKernel<<<dimGrid, dimBlock>>>(dA, dB, dC);

    hipMemcpy(C.data, dC.data, size, hipMemcpyDeviceToHost);

    hipFree(dA.data);
    hipFree(dB.data);
    hipFree(dC.data);

    showMatrix(C);
}

__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C) {
    float value = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < A.width; i++) {
        value += A.data[row * A.width + i] * B.data[i * B.width + col];
    }

    C.data[row * C.width + col] = value;
}

void showMatrix(Matrix m) {
    cout << "size(" << m.height << ", " << m.width << ")" << endl;
    for (int i = 0; i < m.height; i++) {
        for (int j = 0; j < m.width; j++) {
            cout << m.data[i * m.width + j] << "\t";
        }
        cout << endl;
    }
}

int main() {
    Matrix A;
    Matrix B;
    Matrix C;

    A.height = 4;
    A.width = 2;
    A.data = (float *)malloc(A.width * A.height * sizeof(float));
    for (int i = 0; i < A.height * A.width; i++) {
        A.data[i] = i + 1;
    }

    B.height = 2;
    B.width = 4;
    B.data = (float *)malloc(B.width * B.height * sizeof(float));
    for (int i = 0; i < B.height * B.width; i++) {
        B.data[i] = 2 * i;
    }

    C.width = A.height;
    C.height = B.width;
    C.data = (float *)malloc(C.width * C.height * sizeof(float));

    showMatrix(A);
    showMatrix(B);

    MatMaul(A, B, C);
    return 0;
}
