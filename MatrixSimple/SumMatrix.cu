//
// Created by root on 2020/11/11.
//

#include "hip/hip_runtime.h"
#include "iostream"

__global__ void addMatrix(int* a, int* b, int* c, int nx, int ny) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = 0;
    for (; idy < ny; idy++) {
        int index = idy * nx + idx;
        c[index] = a[index] + b[index];
    }
}

int main() {
    int x = 5, y = 2;
    int size = x * y * sizeof(int);

    int *a = (int*) malloc(size);
    int *b = (int*) malloc(size);
    int *c = (int*) malloc(size);

    for (int i = 0; i < x * y; i++) {
        a[i] = i * 2;
        b[i] = i + 1;
    }

    int* h_a;
    int* h_b;
    int* h_c;
    hipMalloc(&h_a, size);
    hipMalloc(&h_b, size);
    hipMalloc(&h_c, size);

    hipMemcpy(h_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(h_b, b, size, hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid((x + block.x - 1) / block.x, (y + block.y - 1) / block.y);

    addMatrix<<<grid, block>>>(h_a, h_b, h_c, x, y);

    hipMemcpy(c, h_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < x * y; i++) {
        std::cout << c[i] << std::endl;
    }

    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
