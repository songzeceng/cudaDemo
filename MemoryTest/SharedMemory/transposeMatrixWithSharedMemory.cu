//
// Created by root on 2020/11/19.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

#define BDIMX 32
#define BDIMY 32

__global__ void transposeNaiveGem(int *in, int *out, int nx, int ny) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < nx && y < ny) {
        out[x * ny + y] = in[y * nx + x];
    }
}

__global__ void transposeNaiveGem2(int *in, int *out, int nx, int ny) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < nx && y < ny) {
        out[y * nx + x] = in[x * ny + y];
        // store is combined but load not
        // Bytes count per request in Pascal seems lower than 128
    }
}

__global__ void transposeSmem(int *in, int *out, int nx, int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x; // thread x coordinate in block(origin matrix)
    int iy = blockIdx.y * blockDim.y + threadIdx.y; // thread y coordinate in block(origin matrix)

    __shared__ int tile[BDIMX][BDIMY];
//    __shared__ int tile[BDIMX][BDIMY + 1]; // We can append one column per row to eliminate store bank conflict

    int ti = iy * nx + ix; // thread data index in matrix / coordinate in origin matrix

    int bidx = threadIdx.y * blockDim.x + threadIdx.x; // thread index in block
    int irow = bidx / blockDim.y; // row of current thread in block
    int icol = bidx % blockDim.y; // column of current thread in block

    ix = blockIdx.y * blockDim.y + icol; // x coordinate in transpose matrix
    iy = blockIdx.x * blockDim.x + irow; // y coordinate in transpose matrix
    int to = iy * ny + ix; // coordinate in transpose matrix

    if (ix < nx && iy < ny) {
        tile[threadIdx.x][threadIdx.y] = in[ti]; // We got bank conflict here, but both the throughput and speed are still higher than global memory
        __syncthreads();
        out[to] = tile[irow][icol]; // Just change the index of data in array
    }
}

__global__ void transposeSmemUnrollPad(int *in, int *out, int nx, int ny) {
    int ix = blockDim.x * blockIdx.x * 2 + threadIdx.x; // thread start x index in block
    int iy = blockDim.y * blockIdx.y + threadIdx.y; // thread start y index in block

    __shared__ int tile[BDIMX * 2][BDIMY + 1];

    int ti = iy * nx + ix; // start data index in block

    int bidx = threadIdx.y * blockDim.x + threadIdx.x; // thread start index in block
    int irow = bidx / blockDim.x; // row in origin matrix
    int icol = bidx % blockDim.x; // column in origin matrix

    int outputIndex = ix * ny + blockIdx.y * BDIMY + irow; // data index in the output array

    if (icol < nx && irow < ny) {
        tile[icol][irow] = in[ti];
        __syncthreads();
        out[outputIndex] = tile[icol][irow];
    }

    if (icol + blockDim.x < nx && irow < ny) {
        tile[icol + blockDim.x][irow] = in[ti + blockDim.x];
        __syncthreads();
        out[outputIndex + blockDim.x * ny] = tile[icol + blockDim.x][irow];
    }
}

int main() {
    int nx = 1024, ny = 1024;
    int nBytes = nx * ny * sizeof(int);

    int *h_in = (int *) malloc(nBytes);
    int *h_out = (int *) malloc(nBytes);

    for (int i = 0; i < nx * ny; i++) {
        h_in[i] = i;
    }

    int *d_in;
    int *d_out;

    hipMalloc(&d_in, nBytes);
    hipMalloc(&d_out, nBytes);

    hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice);

    dim3 blockDim(BDIMX, BDIMY);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);

    transposeNaiveGem<<<gridDim, blockDim>>>(d_in, d_out, nx, ny);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);
//    for (int i = 0; i < nx * ny; i++) {
//        printf("%d->", h_out[i]);
//    }
//
//    printf("\n====================\n");

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);

    dim3 blockDim_(BDIMX, BDIMY);
    dim3 gridDim_((nx + 2 * blockDim_.x - 1) / (2 * blockDim_.x), (ny + blockDim_.y - 1) / blockDim_.y);

    printf("grid_: (%d, %d), block_: (%d, %d)\n", gridDim_.x, gridDim_.y, blockDim_.x, blockDim_.y);
    transposeSmemUnrollPad<<<gridDim_, blockDim_>>>(d_in, d_out, nx, ny);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);
//    for (int i = 0; i < nx * ny; i++) {
//        printf("%d->", h_out[i]);
//    }
//
//    printf("\n====================\n");

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);

    transposeSmem<<<gridDim, blockDim>>>(d_in, d_out, nx, ny);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);
//    for (int i = 0; i < nx * ny; i++) {
//        printf("%d->", h_out[i]);
//    }

    memset(h_out, 0, nBytes);
    hipMemset(d_out, 0, nBytes);

    transposeSmem<<<gridDim, blockDim>>>(d_in, d_out, nx, ny);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);
//    for (int i = 0; i < nx * ny; i++) {
//        printf("%d->", h_out[i]);
//    }

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);
}
