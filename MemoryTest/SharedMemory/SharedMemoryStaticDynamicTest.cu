#include "hip/hip_runtime.h"
#include "stdio.h"

#define BDIMX 32
#define BDIMY 16

#define IPAD 2 // Transactions = BDIMY * sizeof(T) / 8
#define IPAD_D 2

__global__ void setRowReadRow(int* out) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int idx = y * gridDim.x * blockDim.x + x;

    __shared__ int tile[BDIMY][BDIMX];

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();

    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setColReadRow(int* out) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int idx = y * gridDim.x * blockDim.x + x;
    int idxInBlock = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ int tile[BDIMX][BDIMY];

    tile[threadIdx.x][threadIdx.y] = idx;

    __syncthreads();

    out[idx] = tile[idxInBlock / blockDim.y][idxInBlock % blockDim.y];
}

__global__ void setRowReadColPad(int* out) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int idx = y * blockDim.x * gridDim.x + x;
    int idxInBlock = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ int tile[BDIMY][BDIMX + IPAD]; // pad for eliminating bank conflict
    // We had better let the shape of shared memory close to a square.
    // The way of read and write should be specified according to the shape of shared memory

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();

    out[idx] = tile[idxInBlock % blockDim.y][idxInBlock / blockDim.y];
}

// We can see shared memory with pad takes the least time, maybe this is space used for time
__global__ void setRowReadColDyn(int* out) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int idx = y * blockDim.x * gridDim.x + x;

    extern __shared__ int tile[];
    tile[threadIdx.x + threadIdx.y * blockDim.x] = idx;

    __syncthreads();

    int idxInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    int col = idxInBlock / blockDim.y;
    int row = idxInBlock % blockDim.y;

    out[idx] = tile[col + row * blockDim.x];
}

__global__ void setRowReadColDynPad(int* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * blockDim.x * gridDim.x + x;

    extern __shared__ int tile[];

    tile[threadIdx.x + (blockDim.x + IPAD_D) * threadIdx.y] = idx;
    // Now the shared memory`s dim.x needs to be blockDim.x + IPAD
    // Just let the row widder because it`s a one-dim array

    __syncthreads();

    int idxInBlock = threadIdx.y * (blockDim.x) + threadIdx.x;
    int col = idxInBlock / blockDim.y;
    int row = idxInBlock % blockDim.y;

    out[idx] = tile[col + row * (blockDim.x + IPAD_D)];
}

// As we can see, if the coordinates written to and read from the shared memory are same(both x or both y), then the result matrix is the input matrix
// else(write into x and read from y or write into y and read from x) will get the transpose matrix
int main() {
    int x = BDIMX, y = BDIMY;
    int nData = x * y;
    int nBytes = nData * sizeof(int );

    int* outH = (int *) malloc(nBytes);
    int* outD;
    hipMalloc(&outD, nBytes);

    dim3 blockDim(BDIMX, BDIMY);
    dim3 gridDim((x + blockDim.x - 1) / blockDim.x, (y + blockDim.y - 1) / blockDim.y);

    printf("blockDim:(%d, %d), gridDim: (%d, %d)\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);

    setRowReadRow<<<gridDim, blockDim>>>(outD);
    hipDeviceSynchronize();
    hipMemcpy(outH, outD, nBytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < nData; i++) {
        printf("%d->", outH[i]);
    }
    printf("\n\n");
    memset(outH, 0, nData);

    setColReadRow<<<gridDim, blockDim>>>(outD);
    hipDeviceSynchronize();
    hipMemcpy(outH, outD, nBytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < nData; i++) {
        printf("%d->", outH[i]);
    }
    printf("\n\n");
    memset(outH, 0, nData);

    setRowReadColDyn<<<gridDim, blockDim, BDIMX * BDIMY * sizeof(int )>>>(outD);
    hipDeviceSynchronize();
    hipMemcpy(outH, outD, nBytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < nData; i++) {
        printf("%d->", outH[i]);
    }
    printf("\n\n");
    memset(outH, 0, nData);

    setRowReadColPad<<<gridDim, blockDim>>>(outD);
    hipDeviceSynchronize();
    hipMemcpy(outH, outD, nBytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < nData; i++) {
        printf("%d->", outH[i]);
    }
    printf("\n\n");
    memset(outH, 0, nData);

    setRowReadColDynPad<<<gridDim, blockDim, (BDIMX + IPAD_D) * BDIMY * sizeof(int )>>>(outD);
    hipDeviceSynchronize();
    hipMemcpy(outH, outD, nBytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < nData; i++) {
        printf("%d->", outH[i]);
    }
    printf("\n\n");

    hipFree(outD);
    free(outH);

    return 0;
}