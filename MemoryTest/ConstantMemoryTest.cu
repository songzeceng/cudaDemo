//
// Created by root on 2020/11/20.
//

#include "stdio.h"
#include "hip/hip_runtime.h"

#define BDIM 32
#define RADIUS 4

#define a0     0.00000f
#define a1     0.80000f
#define a2    -0.20000f
#define a3     0.03809f
#define a4    -0.00357f

__constant__ float coef[RADIUS + 1];

// constant memory is 64KB for each processor, which is good at uniform read
__global__ void stencil_ld(float *in, float *out) {
    __shared__ float smem[BDIM + 2 * RADIUS];

    int idx = threadIdx.x + blockIdx.x * blockDim.x; // index in global memory

    int sidx = threadIdx.x + RADIUS; // index in shared memory

    smem[sidx] = in[idx]; // thread index + R is the medium data

    if (threadIdx.x < RADIUS) {
        // First four threads get data from thread index(left) and thread index + R + dim(right) into shared memory
        smem[sidx - RADIUS] = in[idx - RADIUS];
        smem[sidx + BDIM] = in[idx + BDIM];
    }

    __syncthreads();

    // calculate stencil
    float tmp = 0.0f;
#pragma unroll
    for (int i = 0; i <= RADIUS; i++) {
        tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    out[idx] = tmp;
}

// restrict memory is 48KB for each processor, which is only suitable for scatter read
__global__ void stencil_ld_readonly(float *in, float *out, float *__restrict__ dcoef) {
    __shared__ float smem[BDIM + 2 * RADIUS];

    int idx = threadIdx.x + blockIdx.x * blockDim.x; // index in global memory

    int sidx = threadIdx.x + RADIUS; // index in shared memory

    smem[sidx] = in[idx]; // thread index + R is the medium data

    if (threadIdx.x < RADIUS) {
        // First four threads get data from thread index(left) and thread index + R + dim(right) into shared memory
        smem[sidx - RADIUS] = in[idx - RADIUS];
        smem[sidx + BDIM] = in[idx + BDIM];
    }

    __syncthreads();

    // calculate stencil
    float tmp = 0.0f;
#pragma unroll
    for (int i = 0; i <= RADIUS; i++) {
        tmp += dcoef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    out[idx] = tmp;
}

void setup_coef() {
    const float h_coef[] = {a0, a1, a2, a3, a4};
    hipMemcpyToSymbol(HIP_SYMBOL(coef), h_coef, (RADIUS + 1) * sizeof(float));
}

int main() {
    int isize = 16;

    size_t nBytes = (isize + 2 * RADIUS) * sizeof(float);

    // allocate host memory
    float *h_in = (float *) malloc(nBytes);
    float *hostRef = (float *) malloc(nBytes);
    float *gpuRef = (float *) malloc(nBytes);

    float *d_in, *d_out, *d_coef;
    hipMalloc((float **) &d_in, nBytes);
    hipMalloc((float **) &d_out, nBytes);
    hipMalloc((float **) &d_coef, (RADIUS + 1) * sizeof(float ));

    for (int i = 0; i < isize + 2 * RADIUS; i++) {
        h_in[i] = (float) i;
    }

    hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice);

    setup_coef();

    dim3 block(BDIM, 1);
    dim3 grid((isize + block.x - 1) / block.x, 1);

    stencil_ld<<<grid, block>>>(d_in + RADIUS, d_out + RADIUS);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < isize + 2 * RADIUS; i++) {
        printf("%f->", gpuRef[i]);
    }
    printf("\n========\n");

    hipMemset(d_out, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    const float h_coef[] = {a0, a1, a2, a3, a4};
    hipMemcpy(d_coef, h_coef, (RADIUS + 1) * sizeof(float ), hipMemcpyHostToDevice);

    stencil_ld_readonly<<<grid, block>>>(d_in + RADIUS, d_out + RADIUS, d_coef);
    hipDeviceSynchronize();
    hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < isize + 2 * RADIUS; i++) {
        printf("%f->", gpuRef[i]);
    }

    return 0;
}