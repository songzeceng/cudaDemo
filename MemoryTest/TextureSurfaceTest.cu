#include "hip/hip_runtime.h"
//
// Created by songzeceng on 2020/11/10.
//

#include "hip/hip_runtime.h"
#include ""
#include "surface_indirect_functions.h"
#include "surface_functions.h"

#include <stdio.h>

void textureObjTest();

void bindTextureRef();

void surfaceObjTest();

void surfaceRefCallKernel();

void surfaceRefBind();

int width = 5, height = 2;
int size = width * height;
float *h_data = (float *) malloc(size * sizeof(float));
texture<float, hipTextureType2D, hipReadModeElementType> texRef;
const surface<void, cudaSurfaceType2D> inputSurf, outputSurf;

__global__ void transformKernel(float *output, hipTextureObject_t texObj, int width, int height, float theta) {
    unsigned int x = blockIdx.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockIdx.y + threadIdx.y;

    float u = x / (float) width - 0.5f;
    float v = y / (float) height - 0.5f;

    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    output[y * width + x] = tex2D<float>(texObj, tu, tv);
}

__global__ void transformKernelRef(float *output, int width, int height, float theta) {
    unsigned int x = blockIdx.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockIdx.y + threadIdx.y;

    float u = x / (float) width - 0.5f;
    float v = y / (float) height - 0.5f;

    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    output[y * width + x] = tex2D(texRef, tu, tv);
}

__global__ void transformKernelSurfaceObj(hipSurfaceObject_t inputSurObj, hipSurfaceObject_t outputSurObj,
                                          int width, int height) {
    unsigned int x = blockIdx.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        uchar4 data;
        surf2Dread(&data, inputSurObj, 4 * x, y);
        surf2Dwrite(data, outputSurObj, 4 * x, y);
    }
}

__global__ void transformKernelSurfaceRef(int width, int height) {
    unsigned int x = blockIdx.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        uchar4 data;
        surf2Dread(&data, inputSurf, 4 * x, y);
        surf2Dwrite(data, outputSurf, 4 * x, y);
    }
}

int main() {
//    textureObjTest();
//    bindTextureRef();
//    surfaceObjTest();
//    surfaceRefBind();

    //    surfaceRefCallKernel();

    return 0;
}

void surfaceRefBind() {
    /////////////////////////////////低级api，绑定到cuda数组//////////////////////////////////////////////
//    const surface<void, cudaSurfaceType2D> surfRef;
//    const surfaceReference* surRefPtr;
//    cudaGetSurfaceReference(&surRefPtr, "surRef");
//    hipChannelFormatDesc desc;
//    hipArray* cuArray;
//    hipGetChannelDesc(&desc, cuArray);
//    cudaBindSurfaceToArray(surfRef, cuArray);

    /////////////////////////////////高级api，绑定到cuda数组//////////////////////////////////////////////
//    hipArray* cuArray;
//    hipChannelFormatDesc des = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
//    hipMallocArray(&cuArray, &des, width, height, hipArraySurfaceLoadStore);
//    surface<void, cudaSurfaceType2D> surfRef;
//    cudaBindSurfaceToArray(surfRef, cuArray);
}

void surfaceRefCallKernel() {
    /////////////////////////////////表面引用调用核函数//////////////////////////////////////////////
    for (int i = 0; i < size; i++) {
        h_data[i] = i;
    }

    hipChannelFormatDesc des = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray* cuInputArray;
    hipMallocArray(&cuInputArray, &des, width, height, hipArraySurfaceLoadStore);
    hipArray* cuOutputArray;
    hipMallocArray(&cuOutputArray, &des, width, height, hipArraySurfaceLoadStore);

    hipMemcpyToArray(cuInputArray, 0, 0, h_data, size * sizeof(float), hipMemcpyHostToDevice);

    cudaBindSurfaceToArray(inputSurf, cuInputArray);
    cudaBindSurfaceToArray(outputSurf, cuOutputArray);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    transformKernelSurfaceRef<<<gridDim, blockDim>>>(width, height);

    float *h_result = (float *) malloc(size * sizeof(float));
    hipMemcpyFromArray(h_result, cuOutputArray, 0, 0, size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        printf("%f\n", h_result[i]);
    }

    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);
}

void surfaceObjTest() {
    for (int i = 0; i < size; i++) {
        h_data[i] = i;
    }

    hipChannelFormatDesc des = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray* cuInputArray;
    hipMallocArray(&cuInputArray, &des, width, height, hipArraySurfaceLoadStore);
    hipArray* cuOutputArray;
    hipMallocArray(&cuOutputArray, &des, width, height, hipArraySurfaceLoadStore);

    hipMemcpyToArray(cuInputArray, 0, 0, h_data, size * sizeof(float), hipMemcpyHostToDevice);

    struct hipResourceDesc resourceDesc;
    memset(&resourceDesc, 0, sizeof(resourceDesc));
    resourceDesc.resType = hipResourceTypeArray;

    resourceDesc.res.array.array = cuInputArray;
    hipSurfaceObject_t cuInputObj = 0, cuOutputObj = 0;
    hipCreateSurfaceObject(&cuInputObj, &resourceDesc);
    resourceDesc.res.array.array = cuOutputArray;
    hipCreateSurfaceObject(&cuOutputObj, &resourceDesc);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    transformKernelSurfaceObj<<<gridDim, blockDim>>>(cuInputObj, cuOutputObj, width, height);

    float *h_result = (float *) malloc(size * sizeof(float));
    hipMemcpyFromArray(h_result, cuOutputArray, 0, 0, size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        printf("%f\n", h_result[i]);
    }

    hipDestroySurfaceObject(cuInputObj);
    hipDestroySurfaceObject(cuOutputObj);
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);
}

void bindTextureRef() {
    /////////////////////////////////低级api，绑定到线性内存//////////////////////////////////////////////
//    texture<float, hipTextureType2D, hipReadModeElementType> texRef;
//    const textureReference *texRefPtr;
//
//    hipGetTextureReference(&texRefPtr, HIP_SYMBOL(&texRef));
//    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
//
//    float *devPtr;
//    size_t pitch;
//    hipMallocPitch((void **) &devPtr, &pitch, width * sizeof(float), height);
//
//    size_t offset;
//    hipBindTexture2D(&offset, texRefPtr, devPtr, &desc, width, height, pitch);

    //////////////////////////////////高级API，绑定到线性内存////////////////////////////////////////////////

//    texture<float, hipTextureType2D, hipReadModeElementType> texRef;
//    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
//
//    float *devPtr;
//    size_t pitch;
//    hipMallocPitch((void **) &devPtr, &pitch, width * sizeof(float), height);
//
//    size_t offset;
//    hipBindTexture2D(&offset, texRef, devPtr, desc, width, height, pitch);

    //////////////////////////////////低级API，绑定到cuda数组////////////////////////////////////////////////
//    texture<float, hipTextureType2D, hipReadModeElementType> texRef;
//    const textureReference* texRefPtr;
//
//    hipGetTextureReference(&texRefPtr, HIP_SYMBOL(&texRef));
//
//    hipChannelFormatDesc desc;
//
//    hipArray *hipArray;
//    hipMallocArray(&hipArray, &desc, width, height);
//
//    hipGetChannelDesc(&desc, hipArray);
//
//    hipBindTextureToArray(texRef, hipArray);

    //////////////////////////////////高级API，绑定到cuda数组////////////////////////////////////////////////
    hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    for (int i = 0; i < size; i++) {
        h_data[i] = i;
    }

    hipArray *hipArray;
    hipMallocArray(&hipArray, &desc, width, height);
    hipMemcpyToArray(hipArray, 0, 0, h_data, size, hipMemcpyHostToDevice);

    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode = hipFilterModeLinear;
    texRef.normalized = 1;

    hipBindTextureToArray(texRef, hipArray);

    float *output;
    hipMalloc(&output, size * sizeof(float));

    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    printf("Ready to call kernel.\n");
    transformKernelRef<<<dimGrid, dimBlock>>>(output, width, height, 0.5f);

    float *h_result = (float *) malloc(size * sizeof(float));
    hipMemcpy(h_result, output, size * sizeof(float), hipMemcpyDeviceToHost);

    printf("Kernel is finished and here`s the result:\n");
    for (int i = 0; i < size; i++) {
        printf("%f\n", h_result[i]);
    }

    hipFreeArray(hipArray);
    hipFree(output);
}

void textureObjTest() {
    for (int i = 0; i < size; i++) {
        h_data[i] = i;
    }

    hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    hipArray *hipArray;

    printf("data initialized.\n");

    hipMallocArray(&hipArray, &desc, width, height);

    hipMemcpyToArray(hipArray, 0, 0, h_data, size, hipMemcpyHostToDevice);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = hipArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    printf("Texture object created.\n");

    float *output;
    hipMalloc(&output, size * sizeof(float));

    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    printf("Ready to call kernel.\n");
    transformKernel<<<dimGrid, dimBlock>>>(output, texObj, width, height, 0.5f);

    float *h_result = (float *) malloc(size * sizeof(float));
    hipMemcpy(h_result, output, size * sizeof(float), hipMemcpyDeviceToHost);

    printf("Kernel is finished and here`s the result:\n");
    for (int i = 0; i < size; i++) {
        printf("%f\n", h_result[i]);
    }

    hipDestroyTextureObject(texObj);
    hipFreeArray(hipArray);
    hipFree(output);
}
