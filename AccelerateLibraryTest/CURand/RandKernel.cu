//
// Created by root on 2020/11/24.
//

#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void device_api_kernel(hiprandState *states, float *out, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // init curand state for each thread
    hiprand_init(9444, tid, 0, states + tid);

    int nthreads = gridDim.x * blockDim.x;
    for (int i = tid; i < N; i += nthreads) {
        float rand = hiprand_uniform(states + tid);
        out[i] = rand * 2;
    }
}

__global__ void host_api_kernel(float *values, float *out, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int nthreads = gridDim.x * blockDim.x;
    for (int i = tid; i < N; i += nthreads) {
        float rand = values[i];
        out[i] = rand * 2;
    }
}

void cuda_device_rand() {
    // use device api to generate random numbers

    static hiprandState *states = NULL;
    static float *dRand = NULL, *hRand = NULL;
    static int dRand_length = 1000000;

    int block = 256;
    int grid = 30;

    hipMalloc(&dRand, sizeof(float) * dRand_length);
    hipMalloc(&states, sizeof(hiprandState) * block * grid);
    hRand = (float *) malloc(sizeof(float) * dRand_length);
    device_api_kernel<<<grid, block>>>(states, dRand, dRand_length);

    hipMemcpy(hRand, dRand, sizeof(float) * dRand_length, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%.2f\t", hRand[i]);
    }
    printf("\n");

    free(hRand);
    hipFree(dRand);
}

void cuda_host_rand() {
    // generate random data with host api

    static hiprandGenerator_t randGen;
    static float *dRand = NULL, *dOut, *hOut;
    static int dRand_length = 1000000, dRand_used = 1000000;

    hipMalloc(&dRand, sizeof(float) * dRand_length);
    hipMalloc(&dOut, sizeof(float) * dRand_length);
    hOut = (float *) malloc(sizeof(float) * dRand_length);
    hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);

    hiprandGenerateUniform(randGen, dRand, dRand_length); // the new data are in device memory

    host_api_kernel<<<30, 256>>>(dRand, dOut, dRand_length);

    hipMemcpy(hOut, dOut, sizeof(float) * dRand_length, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%.2f\t", hOut[i]);
    }
    printf("\n");

    free(hOut);
    hipFree(dOut);
    hipFree(dRand);
}

// nvcc -lcurand RandKernel.cu -o RandKernel
int main() {
    printf("host:\n");
    cuda_host_rand();

    printf("device:\n");
    cuda_device_rand();
    return 0;
}