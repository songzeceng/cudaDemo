//
// Created by root on 2020/11/24.
//

#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void initialize_state(hiprandState* states) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // init curand state for each thread
    hiprand_init(9444, tid, 0, states + tid);
}

__global__ void refill_randoms(float *dRand, int N, hiprandState* states) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState *state = states + tid;

    for (int i = tid; i < N; i += nthreads) {
        dRand[i] = hiprand_uniform(state);
        // generate random number following uniform distribution for each thread
        // the number of random numbers is N in total
    }
}

float cuda_device_rand() {
    // use device api to generate random numbers

    static hiprandState *states = NULL;
    static float *dRand = NULL, *hRand = NULL;
    static int dRand_length = 1000000, dRand_used = 1000000;

    int block = 256;
    int grid = 30;

    if (dRand == NULL) {
        // if dRand is null, then allocate memory and initialize states
        hipMalloc(&dRand, sizeof(float ) * dRand_length);
        hipMalloc(&states, sizeof(hiprandState) * block * grid);
        hRand = (float *) malloc(sizeof(float ) * dRand_length);
        initialize_state<<<grid, block>>>(states);
    }

    if (dRand_used == dRand_length) {
        // if all random data have been traversed, we should generate a new batch of data
        refill_randoms<<<grid, block>>>(dRand, dRand_length, states);
        hipMemcpy(hRand, dRand, sizeof(float ) * dRand_length, hipMemcpyDeviceToHost);
        dRand_used = 0;
    }
    return hRand[dRand_used++];
}

float cuda_host_rand() {
    // generate random data with host api

    static hiprandGenerator_t randGen;
    static float *dRand = NULL, *hRand = NULL;
    static int dRand_length = 1000000, dRand_used = 1000000;

    if (dRand == NULL) {
        // if dRand is null, then allocate memory and create generator
        hipMalloc(&dRand, sizeof(float ) * dRand_length);
        hRand = (float *) malloc(sizeof(float ) * dRand_length);
        hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
    }

    if (dRand_used == dRand_length) {
        // if all random data generated have been traversed, we should generate a batch of new data
        hiprandGenerateUniform(randGen, dRand, dRand_length); // the new data are in device memory
        hipMemcpy(hRand, dRand, sizeof(float ) * dRand_length, hipMemcpyDeviceToHost);
        dRand_used = 0;
    }
    return hRand[dRand_used++];
}

// nvcc -lcurand ReplaceRand.cu -o ReplaceRand
int main() {
    for (int i = 0; i < 256; i++) {
        float h = cuda_host_rand();
        float d = cuda_device_rand();
        printf("h = %.2f, d = %.2f\n", h, d);
    }
    return 0;
}