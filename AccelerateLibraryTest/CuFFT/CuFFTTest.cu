//
// Created by root on 2020/11/24.
//

#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "stdio.h"

int nprint = 30;

void generate_fake_samples(int N, float **out) {
    float *result = (float *) malloc(sizeof(float ) * N);
    double delta = M_PI / 20.0;

    for (int i = 0; i < N; i++) {
        result[i] = cos(i * delta);
    }

    *out = result;
}

void real_to_complex(float *r, hipfftComplex **complex, int N) {
    (*complex) = (hipfftComplex *) malloc(sizeof(hipfftComplex) * N);

    for (int i = 0; i < N; i++) {
        (*complex)[i].x = r[i];
        (*complex)[i].y = 0;
    }
}

// nvcc -lcufft CuFFTTest.cu -o CuFFTTest
int main() {
    int N = 2048;
    float *samples;
    hipfftHandle plan;
    hipfftComplex *dComplexSamples, *complexSamples, *complexFreq;

    // allocate memory
    samples = (float *) malloc(N * sizeof(float ));
    complexSamples = (hipfftComplex *) malloc(N * sizeof(hipfftComplex));
    complexFreq = (hipfftComplex *) malloc(N * sizeof(hipfftComplex));
    hipMalloc(&dComplexSamples, sizeof(hipfftComplex) * N);

    // generate input
    generate_fake_samples(N, &samples);
    printf("Initial samples:\n");
    for (int i = 0; i < nprint; i++) {
        printf("%.2f\t", samples[i]);
    }
    printf("\n");
    real_to_complex(samples, &complexSamples, N);

    // create cufft plan with type complex to complex
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);

    // copy data to device
    hipMemcpy(dComplexSamples, complexSamples, N * sizeof(hipfftComplex ), hipMemcpyHostToDevice);

    // execute forward fourier transform
    hipfftExecC2C(plan, dComplexSamples, dComplexSamples, HIPFFT_FORWARD);

    // get data from device and print demo data
    hipMemcpy(complexFreq, dComplexSamples, sizeof(hipfftComplex) * N, hipMemcpyDeviceToHost);
    printf("Fourier coefficient:\n");
    for (int i = 0; i < nprint; i++) {
        printf("(%.2f, %.2f)\t", complexFreq[i].x , complexFreq[i].y);
    }
    printf("\n");

    free(samples);
    free(complexSamples);
    free(complexFreq);
    hipFree(dComplexSamples);
    hipfftDestroy(plan);

    return 0;
}