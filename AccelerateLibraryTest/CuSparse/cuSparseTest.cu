//
// Created by root on 2020/11/23.
//

#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipsparse.h"

int M = 1024;
int N = 1024;

void generate_random_vector(int n, float** outX) {
    int i = 0;
    double rMax = (double ) RAND_MAX;
    float* X = (float *) malloc(n * sizeof(float ));
    for (; i < n; i++) {
        X[i] = ((double ) rand()) / rMax * 100.0;
    }
    *outX = X;
}

int generate_random_dense_matrix(int m, int n, float** outX) {
    int i = 0, j = 0;
    double rMax = (double ) RAND_MAX;
    float *A = (float *) malloc(m * n * sizeof(float ));
    int totalNnz = 0;

    for (; j < n; j++) {
        for (; i < m; i++) {
            int r = rand();
            float* curr = A + (j * m + i);
            if (r % 3 > 0) {
                *curr = 0.0f;
            } else {
                *curr = ((double ) r) / rMax * 100;
            }

            if (*curr != 0.0f) {
                totalNnz++;
            }
        }
    }
    *outX = A;
    return totalNnz;
}

//  nvcc -lcusparse -o cusparse cuSparseTest.cu
int main() {
    int totalNnz;
    float *A, *dA, *dCsrValAm, *X, *Y, *dX, *dY;
    int *dNnzPerRow, *dCsrRowPtrA, *dCsrColIndA;
    float alpha = 3.0f, beta = 4.0f;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    // Allocate host memory
    A = (float *) malloc(sizeof(float ) * M * N);
    X = (float *) malloc(sizeof(float ) * N);
    Y = (float *) malloc(sizeof(float ) * M);

    // generate input
    srand(3325);
    int trueNnz = generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    generate_random_vector(M, &Y);

    // create cusparse handle
    hipsparseCreate(&handle);

    // allocate device memory
    hipMalloc(&dX, sizeof(float ) * N);
    hipMalloc(&dY, sizeof(float ) * M);
    hipMalloc(&dA, sizeof(float ) * M * N);
    hipMalloc(&dNnzPerRow, sizeof(float ) * M);

    // create matrix descriptor
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    // copy data to device
    hipMemcpy(dX, X, sizeof(float ) * N, hipMemcpyHostToDevice);
    hipMemcpy(dY, Y, sizeof(float ) * M, hipMemcpyHostToDevice);
    hipMemcpy(dA, A, sizeof(float ) * M * N, hipMemcpyHostToDevice);

    // compute the number of non-zero elements in each row of A
    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, descr, dA, M, dNnzPerRow, &totalNnz);

    if (totalNnz == trueNnz) {
        printf("Non-zero element count matches which is %d!\n", totalNnz);
    }

    // allocate memory for csr vectors
    hipMalloc(&dCsrValAm, sizeof(float ) * totalNnz);
    hipMalloc(&dCsrRowPtrA, sizeof(int ) * (M + 1));
    hipMalloc(&dCsrColIndA, sizeof(int ) * totalNnz);

    // get csr vectors from matrix in device memory
    hipsparseSdense2csr(handle, M, N, descr, dA, M, dNnzPerRow, dCsrValAm, dCsrRowPtrA, dCsrColIndA);

    // y = alpha * op(A) * x  + beta * y, where op is non-transpose here.
    hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M, N, totalNnz, &alpha, descr,
                   dCsrValAm, dCsrRowPtrA, dCsrColIndA, dX, &beta, dY);

    // get result from device and show first 10 of it
    hipMemcpy(Y, dY, sizeof(float ) * M, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%.2f\t", Y[i]);
    }
    printf("\n");

    // free resource
    free(A);
    free(X);
    free(Y);
    hipFree(dX);
    hipFree(dY);
    hipFree(dA);
    hipFree(dNnzPerRow);
    hipFree(dCsrValAm);
    hipFree(dCsrRowPtrA);
    hipFree(dCsrColIndA);

    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(handle);

    return 0;
}
