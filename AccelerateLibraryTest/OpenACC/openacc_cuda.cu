//
// Created by root on 2020/11/24.
//

#include "stdio.h"
#include "stdlib.h"
#include "hiprand/hiprand.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"

#define M 10
#define N 10
#define P 10

// nvcc openacc_cuda.cu -o openacc_cuda -lcurand -lcublas
int main() {
    float *__restrict__ d_A, *__restrict__ d_B, *__restrict__ d_C;
    float *d_row_sums;
    float total_sum;
    hiprandGenerator_t rand_state;
    hipblasHandle_t cublas_handle;

    hiprandCreateGenerator(&rand_state, HIPRAND_RNG_PSEUDO_DEFAULT);
    hipblasCreate(&cublas_handle);
    printf("hiprand and cublas created success!\n");

    hipMalloc((void **) &d_A, M * N * sizeof(float));
    hipMalloc((void **) &d_B, N * P * sizeof(float));
    hipMalloc((void **) &d_C, M * P * sizeof(float));
    hipMalloc((void **) &d_row_sums, M * sizeof(float ));
    printf("Memory allocated successfully on device!\n");

    hiprandGenerateUniform(rand_state, d_A, M * N);
    hiprandGenerateUniform(rand_state, d_B, N * P);
    printf("Data initialized successfully on device!\n");

#pragma acc parallel loop gang deviceptr(d_A, d_B, d_C) // cannot put device data directly into openacc area for openacc 20.7
    {
        for (int i = 0; i < M; i++) { // A row
#pragma acc loop worker vector
            {
                for (int j = 0; j < P; j++) { // B col
                    float sum = 0.0f;
                    for (int k = 0; k < N; k++) { // A col and B row
//                        printf("sum for (%d, %d) at %d\n", i, j, k);
                        sum += d_A[i * N + k] * d_B[k * P + j]; // segmentation fault here
                    }
                    d_C[i * P + j] = sum;
                }
            }
        }
    }
    printf("Matrix multiplication and add performed successfully!\n");

    hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE);

    for (int i = 0; i < M; i++) {
        hipblasSasum(cublas_handle, P, d_C + i * P, 1, d_row_sums + i);
    }

    hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST);

    hipblasSasum(cublas_handle, M, d_row_sums, 1, &total_sum);
    hipDeviceSynchronize();
    printf("cublas sum for each row successfully!\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    printf("Memory free successfully on device!\n");

    printf("sum = %.4f\n", total_sum);

    return 0;
}