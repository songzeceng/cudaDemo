//
// Created by root on 2020/11/24.
//

#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

int M = 1024;
int N = 1024;

void generate_random_vector(int n, float** outX) {
    int i = 0;
    double rMax = (double ) RAND_MAX;
    float* X = (float *) malloc(n * sizeof(float ));
    for (; i < n; i++) {
        X[i] = ((double ) rand()) / rMax * 100.0;
    }
    *outX = X;
}

void generate_random_dense_matrix(int m, int n, float** outX) {
    int i = 0, j = 0;
    double rMax = (double ) RAND_MAX;
    float *A = (float *) malloc(m * n * sizeof(float ));

    for (; j < n; j++) {
        for (; i < m; i++) {
            int r = rand();
            float* curr = A + (j * m + i);
            if (r % 3 > 0) {
                *curr = 0.0f;
            } else {
                *curr = ((double ) r) / rMax * 100;
            }
        }
    }
    *outX = A;
}

//  nvcc -lcublas -o CuBlasTest CuBlasTest.cu
int main() {
    int i;
    float *A, *dA, *X, *dX, *Y, *dY;
    float alpha = 3.0f, beta=4.0f;
    hipblasHandle_t handle;

    // allocate memory
    A = (float *) malloc(sizeof(float ) * M * N);
    X = (float *) malloc(sizeof(float ) * N);
    Y = (float *) malloc(sizeof(float ) * M);
    hipMalloc(&dA, sizeof(float ) * M * N);
    hipMalloc(&dX, sizeof(float ) * N);
    hipMalloc(&dY, sizeof(float) * M);

    // generate input
    srand(3432);
    generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    generate_random_vector(M, &Y);

    // create cublas handle
    hipblasCreate(&handle);

    // get cublas data from original input
    hipblasSetMatrix(M, N, sizeof(float), A, M, dA, N);
    hipblasSetVector(N, sizeof(float ), X, 1, dX, 1);
    hipblasSetVector(M, sizeof(float ), Y, 1, dY, 1);

    // perform matrix multiplication y = alpha * op(A) * x  + beta * y
    hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, dA, M, dX, 1, &beta, dY, 1);

    // get result from cublas and print demo data
    hipblasGetVector(M, sizeof(float ), dY, 1, Y, 1);

    for (i = 0; i < 10; i++) {
        printf("%.2f\t", Y[i]);
    }
    printf("\n");

    // free memory
    free(A);
    free(X);
    free(Y);
    hipFree(dA);
    hipFree(dX);
    hipFree(dY);
    return 0;
}