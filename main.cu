#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MyKernel(float *dev0Ptr, float *dev1Ptr, int begin, int size) {
    for (int i = begin; i < begin + size; i++) {
        dev1Ptr[i] = dev0Ptr[i] * 2;
    }
}

void CUDART_CB MyCallback(hipStream_t stream, hipError_t status, void *data) {
    printf("Inside callback %d with status %d\n", (int) data, status);
}

int main() {

    hipStream_t stream[2];
    for (int i = 0; i < 2; i++) {
        hipStreamCreate(&stream[i]);
    }

//    int priority_high, priority_low;
//    hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
//
//    hipStream_t st_high, st_low;
//    hipStreamCreateWithPriority(&st_high, hipStreamNonBlocking, priority_high);
//    hipStreamCreateWithPriority(&st_low, hipStreamNonBlocking, priority_low);

    float *hostPtr;
    int size = 16;
    hipHostMalloc(&hostPtr, 2 * size * sizeof(float ));

    float *dev0Ptr;
    hipMalloc(&dev0Ptr, 2 * size * sizeof(float ));
    float *dev1Ptr;
    hipMalloc(&dev1Ptr, 2 * size * sizeof(float ));

    for (int i = 0; i < 2 * size; i++) {
        hostPtr[i] = i + 1;
    }

//    for (int i = 0; i < 2; i++) {
//        hipMemcpyAsync(&dev0Ptr[i * size], &hostPtr[i * size], size * sizeof(float ), hipMemcpyHostToDevice, stream[i]);
//
//        MyKernel<<<100, 512, 0, stream[i]>>>(dev0Ptr, dev1Ptr, i * size, size);
//
//        hipMemcpyAsync(&hostPtr[i * size], &dev1Ptr[i * size], size * sizeof(float ), hipMemcpyDeviceToHost, stream[i]);
//
//        hipStreamAddCallback(stream[i], MyCallback, (void *) i, 0);
//
//        hipStreamDestroy(stream[i]);
//    }
//
//    for (int i = 0; i < 2 * size; i++) {
//        printf("%f\t", hostPtr[i]);
//    }
//    printf("\n");

///////////////////////////////////////////////////////////

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

//    hipEventDestroy(start);
//    hipEventDestroy(stop);

    hipEventRecord(start, 0);
    for (int i = 0; i < 2; ++i) {
        hipMemcpyAsync(dev0Ptr + i * size, hostPtr + i * size,
                        size * sizeof(float ), hipMemcpyHostToDevice, stream[i]);
        MyKernel<<<1, 1, 0, stream[i]>>>
                (dev0Ptr, dev1Ptr, i * size, size);
        hipMemcpyAsync(hostPtr + i * size, dev1Ptr + i * size,
                        size * sizeof(float ), hipMemcpyDeviceToHost, stream[i]);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "time: " << elapsedTime << std::endl;

    for (int i = 0; i < 2 * size; i++) {
        printf("%f\t", hostPtr[i]);
    }
    printf("\n");

    ///////////////////////////////////////////////////////

//    for (int i = 0; i < 2; i++) {
//        hipMemcpyAsync(dev0Ptr + i * size, hostPtr + i * size, size, hipMemcpyHostToDevice, stream[i]);
//    }
//
//    for (int i = 0; i < 2; i++) {
//        MyKernel<<<100, 512, 0, stream[i]>>>(dev0Ptr + i * size, dev1Ptr + i * size, size);
//    }
//
//    for (int i = 0; i < 2; i++) {
//        hipMemcpyAsync(hostPtr + i * size, dev1Ptr + i * size, size, hipMemcpyDeviceToHost, stream[i]);
//    }

    return 0;
}
