#include "hip/hip_runtime.h"
//
// Created by songzeceng on 2020/11/13.
// 编译：nvcc DynamicParallelTest.cu -gencode arch=compute_61,code=sm_61 -rdc=true -o DynamicParallelTest

#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ void printArray(int *data, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d\t", data[i]);
    }
}

__global__ void child_launch(int *data, int n) {
//    data[threadIdx.x] = data[threadIdx.x] + 1000;
    printf("Hello ");
}

__global__ void parent_launch(int *data, int n) {
//    data[threadIdx.x] = threadIdx.x;
//
//    __syncthreads();
//
//    printArray(data, n);
//
//    if (threadIdx.x == 0) {
//        child_launch<<<1, 6>>>(data, n);
//    }
//#endif

    child_launch<<<1, 1>>>(data, n);
    hipDeviceSynchronize();

    printf(" World!\n");

    int *d = (int *) malloc(10 * sizeof(int));
    int *a = (int *) malloc(20 * sizeof(int));
    int *a_1;
    hipMalloc(&a_1, 50 * sizeof(int));
    int *a_2;
    hipMalloc(&a_2, 90 * sizeof(int));

    free(d);
    free(a);
    hipFree(a_1);
    hipFree(a_2);

//    printArray(data, n);
//
//    __syncthreads();
}

int main() {
    hipSetDevice(0);

    int size = 6;
    int *h_data = (int *) malloc(size * sizeof(int));
    int *d_data;

    for (int i = 0; i < size; i++) {
        h_data[i] = 0;
    }
    hipMalloc(&d_data, size * sizeof(int));
    hipMemcpy(d_data, h_data, size * sizeof(int), hipMemcpyHostToDevice);

    parent_launch<<<1, 1>>>(d_data, size);
    hipDeviceSynchronize();

    hipFree(d_data);
    free(h_data);
    return 0;
}
