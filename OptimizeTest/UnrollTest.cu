//
// Created by root on 2020/11/12.
//

#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void unrollTestKernel(int *count) {
#pragma unroll 4
    for (int i = 0; i < 20; i++) {
        (*count)++;
    }
}

int main() {
    int *n_h = (int *) malloc(sizeof(int ) );
    *n_h = 0;
    int *h_d;
    hipMalloc(&h_d, sizeof(int ));
    hipMemcpy(h_d, n_h, sizeof(int ), hipMemcpyHostToDevice);

    unrollTestKernel<<<1, 1>>>(h_d);

    hipMemcpy(n_h, h_d, sizeof(int ), hipMemcpyDeviceToHost);

    printf("count = %d\n", *n_h); // 20

    hipFree(h_d);
    free(n_h);

    return 0;
}
